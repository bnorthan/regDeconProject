#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
// Includes CUDA
//#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//
#include <memory.h>
#include ""
#include <hipfft/hipfft.h>
//#include <hipfft/hipfftw.h> // ** cuFFT also comes with CPU-version FFTW, but seems not to work when image size is large.
#include "fftw3.h"


#include "cukernel.cuh"
extern "C" {
#include "powell.h"
}
#include "apifunc_internal.h"

#define SMALLVALUE 0.01
#define NDIM 12
hipError_t cudaStatus;
#define cudaCheckErrors(msg) \
    do { \
        cudaStatus = hipGetLastError(); \
        if (cudaStatus != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(cudaStatus), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
								        } \
				    } while (0)

extern "C"
bool isPow2(int x)
{
	return ((x&(x - 1)) == 0);
};

//Round a / b to nearest higher integer value
inline long long int iDivUp(long long int a, long long int b)
{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Align a to nearest higher multiple of b
inline long long int iAlignUp(long long int a, long long int b)
{
	return (a % b != 0) ? (a - a % b + b) : a;
}

int snapTransformSize(int dataSize)//
{
	int hiBit;
	unsigned int lowPOT, hiPOT;

	dataSize = iAlignUp(dataSize, 16);

	for (hiBit = 31; hiBit >= 0; hiBit--)
		if (dataSize & (1U << hiBit))
		{
			break;
		}

	lowPOT = 1U << hiBit;

	if (lowPOT == (unsigned int)dataSize)
	{
		return dataSize;
	}

	hiPOT = 1U << (hiBit + 1);

	if (hiPOT <= 128)
	{
		return hiPOT;
	}
	else
	{
		return iAlignUp(dataSize, 64);
	}
}

//////////////// Basic math functions  /////////////////
// CPU functions
// sum
template <class T>
double sumcpu(T *h_idata, size_t totalSize) {
	double sumValue = 0;
	for (size_t i = 0; i < totalSize; i++) {
		sumValue += (double)h_idata[i];
	}
	return sumValue;
}
template double sumcpu<int>(int *h_idata, size_t totalSize);
template double sumcpu<float>(float *h_idata, size_t totalSize);
template double sumcpu<double>(double *h_idata, size_t totalSize);
// add
template <class T>
void addcpu(T *h_odata, T *h_idata1, T *h_idata2, size_t totalSize){
	for (size_t i = 0; i < totalSize; i++)
		h_odata[i] = h_idata1[i] + h_idata2[i];
}
template void addcpu<int>(int *h_odata, int *h_idata1, int *h_idata2, size_t totalSize);
template void addcpu<float>(float *h_odata, float *h_idata1, float *h_idata2, size_t totalSize);
template void addcpu<double>(double *h_odata, double *h_idata1, double *h_idata2, size_t totalSize);

template <class T>
void addvaluecpu(T *h_odata, T *h_idata1, T h_idata2, size_t totalSize){
	const T b = h_idata2;
	for (size_t i = 0; i < totalSize; i++)
		h_odata[i] = h_idata1[i] + b;
}
template void addvaluecpu<int>(int *h_odata, int *h_idata1, int h_idata2, size_t totalSize);
template void addvaluecpu<float>(float *h_odata, float *h_idata1, float h_idata2, size_t totalSize);
template void addvaluecpu<double>(double *h_odata, double *h_idata1, double h_idata2, size_t totalSize);
// subtract
template <class T>
void subcpu(T *h_odata, T *h_idata1, T *h_idata2, size_t totalSize){
	for (size_t i = 0; i < totalSize; i++)
		h_odata[i] = h_idata1[i] - h_idata2[i];
}
template void subcpu<int>(int *h_odata, int *h_idata1, int *h_idata2, size_t totalSize);
template void subcpu<float>(float *h_odata, float *h_idata1, float *h_idata2, size_t totalSize);
template void subcpu<double>(double *h_odata, double *h_idata1, double *h_idata2, size_t totalSize);
// multiply
template <class T>
void multicpu(T *h_odata, T *h_idata1, T *h_idata2, size_t totalSize){
	for (size_t i = 0; i < totalSize; i++)
		h_odata[i] = h_idata1[i] * h_idata2[i];
}
template void multicpu<int>(int *h_odata, int *h_idata1, int *h_idata2, size_t totalSize);
template void multicpu<float>(float *h_odata, float *h_idata1, float *h_idata2, size_t totalSize);
template void multicpu<double>(double *h_odata, double *h_idata1, double *h_idata2, size_t totalSize);
//divide
template <class T>
void divcpu(T *h_odata, T *h_idata1, T *h_idata2, size_t totalSize){
	for (size_t i = 0; i < totalSize; i++)
		h_odata[i] = h_idata1[i] / h_idata2[i];
}
template void divcpu<int>(int *h_odata, int *h_idata1, int *h_idata2, size_t totalSize);
template void divcpu<float>(float *h_odata, float *h_idata1, float *h_idata2, size_t totalSize);
template void divcpu<double>(double *h_odata, double *h_idata1, double *h_idata2, size_t totalSize);

template <class T>
void multivaluecpu(T *h_odata, T *h_idata1, T h_idata2, size_t totalSize){
	for (size_t i = 0; i < totalSize; i++)
		h_odata[i] = h_idata1[i] * h_idata2;
}
template void multivaluecpu<int>(int *h_odata, int *h_idata1, int h_idata2, size_t totalSize);
template void multivaluecpu<float>(float *h_odata, float *h_idata1, float h_idata2, size_t totalSize);
template void multivaluecpu<double>(double *h_odata, double *h_idata1, double h_idata2, size_t totalSize);

extern "C"
void multicomplexcpu(fComplex *h_odata, fComplex *h_idata1, fComplex *h_idata2, size_t totalSize){
	fComplex a;
	fComplex b;
	for (size_t i = 0; i < totalSize; i++){
		a = h_idata1[i];
		b = h_idata2[i];
		h_odata[i].x = a.x*b.x - a.y*b.y;
		h_odata[i].y = a.x*b.y + a.y*b.x;
	}		
}

// max3Dcpu: find max value and coordinates
template <class T>
T max3Dcpu(size_t *corXYZ, T *h_idata, size_t sx, size_t sy, size_t sz) {
	T peakValue = h_idata[0];
	T t;
	size_t sx0 = 0, sy0 = 0, sz0 = 0;
	for (size_t i = 0; i < sx; i++) {
		for (size_t j = 0; j < sy; j++) {
			for (size_t k = 0; k < sz; k++) {
				t = h_idata[i + j * sx + k * sx * sy];
				if (peakValue < t) {
					peakValue = t;
					sx0 = i;
					sy0 = j;
					sz0 = k;
				}
			}
		}
	}

	corXYZ[0] = sx0; corXYZ[1] = sy0; corXYZ[2] = sz0;
	return peakValue;
}
template int max3Dcpu<int>(size_t *corXYZ, int *h_idata, size_t sx, size_t sy, size_t sz);
template float max3Dcpu<float>(size_t *corXYZ, float *h_idata, size_t sx, size_t sy, size_t sz);
template double max3Dcpu<double>(size_t *corXYZ, double *h_idata, size_t sx, size_t sy, size_t sz);

// max with a single value
template <class T>
void maxvaluecpu(T *h_odata, T *h_idata1, T h_idata2, size_t totalSize) {
	T a;
	const T b = h_idata2;
	for (size_t i = 0; i < totalSize; i++) {
		a = h_idata1[i];
		h_odata[i] = (a > b) ? a : b;
	}
}
template void maxvaluecpu<int>(int *d_odata, int *d_idata1, int d_idata2, size_t totalSize);
template void maxvaluecpu<float>(float *d_odata, float *d_idata1, float d_idata2, size_t totalSize);
template void maxvaluecpu<double>(double *d_odata, double *d_idata1, double d_idata2, size_t totalSize);

template <class T>
void changestorageordercpu(T *h_odata, T *h_idata, size_t sx, size_t sy, size_t sz, int orderMode) {
	//orderMode
	// 1: change tiff storage order to C storage order
	//-1: change C storage order to tiff storage order
	if (orderMode == 1) {
		for (size_t i = 0; i < sx; i++) {
			for (size_t j = 0; j < sy; j++) {
				for (size_t k = 0; k < sz; k++) {
					h_odata[i*sy*sz + j*sz + k] = h_idata[k*sy*sx + j*sx + i];
				}
			}
		}
	}
	else if (orderMode == -1) {//change C storage order to tiff storage order:
		for (size_t i = 0; i < sx; i++) {
			for (size_t j = 0; j < sy; j++) {
				for (size_t k = 0; k < sz; k++) {
					h_odata[k*sy*sx + j*sx + i] = h_idata[i*sy*sz + j*sz + k];
				}
			}
		}
	}
}
template void changestorageordercpu<int>(int *h_odata, int *h_idata, size_t sx, size_t sy, size_t sz, int orderMode);
template void changestorageordercpu<float>(float *h_odata, float *h_idata, size_t sx, size_t sy, size_t sz, int orderMode);
template void changestorageordercpu<double>(double *h_odata, double *h_idata, size_t sx, size_t sy, size_t sz, int orderMode);


///// GPU functions
//add
template <class T>
void add3Dgpu(T *d_odata, T *d_idata1, T *d_idata2, size_t sx, size_t sy, size_t sz){
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sx, blockSize3Dx), iDivUp(sy, blockSize3Dy), iDivUp(sz, blockSize3Dz));
	add3Dkernel<T> <<<grids, threads>>>(d_odata, d_idata1, d_idata2, sx, sy, sz);
	hipDeviceSynchronize();
}
template void add3Dgpu<int>(int *d_odata, int *d_idata1, int *d_idata2, size_t sx, size_t sy, size_t sz);
template void add3Dgpu<float>(float *d_odata, float *d_idata1, float *d_idata2, size_t sx, size_t sy, size_t sz);
template void add3Dgpu<double>(double *d_odata, double *d_idata1, double *d_idata2, size_t sx, size_t sy, size_t sz);

// add with a single value
template <class T>
void addvaluegpu(T *d_odata, T *d_idata1, T d_idata2, size_t sx, size_t sy, size_t sz){
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sx, blockSize3Dx), iDivUp(sy, blockSize3Dy), iDivUp(sz, blockSize3Dz));
	addvaluekernel<T> <<<grids, threads >>>(d_odata, d_idata1, d_idata2, sx, sy, sz);
	hipDeviceSynchronize();
}
template void addvaluegpu<int>(int *d_odata, int *d_idata1, int d_idata2, size_t sx, size_t sy, size_t sz);
template void addvaluegpu<float>(float *d_odata, float *d_idata1, float d_idata2, size_t sx, size_t sy, size_t sz);
template void addvaluegpu<double>(double *d_odata, double *d_idata1, double d_idata2, size_t sx, size_t sy, size_t sz);

//subtract
template <class T>
void sub3Dgpu(T *d_odata, T *d_idata1, T *d_idata2, size_t sx, size_t sy, size_t sz){
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sx, blockSize3Dx), iDivUp(sy, blockSize3Dy), iDivUp(sz, blockSize3Dz));
	sub3Dkernel<T> <<<grids, threads>>>(d_odata, d_idata1, d_idata2, sx, sy, sz);
	hipDeviceSynchronize();
}
template void sub3Dgpu<int>(int *d_odata, int *d_idata1, int *d_idata2, size_t sx, size_t sy, size_t sz);
template void sub3Dgpu<float>(float *d_odata, float *d_idata1, float *d_idata2, size_t sx, size_t sy, size_t sz);
template void sub3Dgpu<double>(double *d_odata, double *d_idata1, double *d_idata2, size_t sx, size_t sy, size_t sz);


//multiply
template <class T>
void multi3Dgpu(T *d_odata, T *d_idata1, T *d_idata2, size_t sx, size_t sy, size_t sz){
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sx, blockSize3Dx), iDivUp(sy, blockSize3Dy), iDivUp(sz, blockSize3Dz));
	multi3Dkernel<T> <<<grids, threads>>>(d_odata, d_idata1, d_idata2, sx, sy, sz);
	hipDeviceSynchronize();
}
template void multi3Dgpu<int>(int *d_odata, int *d_idata1, int *d_idata2, size_t sx, size_t sy, size_t sz);
template void multi3Dgpu<float>(float *d_odata, float *d_idata1, float *d_idata2, size_t sx, size_t sy, size_t sz);
template void multi3Dgpu<double>(double *d_odata, double *d_idata1, double *d_idata2, size_t sx, size_t sy, size_t sz);

// multiply with a single value
template <class T>
void multivaluegpu(T *d_odata, T *d_idata1, T d_idata2, size_t sx, size_t sy, size_t sz){
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sx, blockSize3Dx), iDivUp(sy, blockSize3Dy), iDivUp(sz, blockSize3Dz));
	multivaluekernel<T> <<<grids, threads>>>(d_odata, d_idata1, d_idata2, sx, sy, sz);
	hipDeviceSynchronize();
}
template void multivaluegpu<int>(int *d_odata, int *d_idata1, int d_idata2, size_t sx, size_t sy, size_t sz);
template void multivaluegpu<float>(float *d_odata, float *d_idata1, float d_idata2, size_t sx, size_t sy, size_t sz);
template void multivaluegpu<double>(double *d_odata, double *d_idata1, double d_idata2, size_t sx, size_t sy, size_t sz);

//multiply float complex
extern "C"
void multicomplex3Dgpu(fComplex *d_odata, fComplex *d_idata1, fComplex *d_idata2, size_t sx, size_t sy, size_t sz){
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sx, blockSize3Dx), iDivUp(sy, blockSize3Dy), iDivUp(sz, blockSize3Dz));
	multicomplex3Dkernel<<<grids, threads>>>(d_odata, d_idata1, d_idata2, sx, sy, sz);
	hipDeviceSynchronize();
}

//multiply float complex and do normalization
extern "C"
void multicomplexnorm3Dgpu(fComplex *d_odata, fComplex *d_idata1, fComplex *d_idata2, size_t sx, size_t sy, size_t sz){
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sx, blockSize3Dx), iDivUp(sy, blockSize3Dy), iDivUp(sz, blockSize3Dz));
	multicomplexnorm3Dkernel <<<grids, threads>>>(d_odata, d_idata1, d_idata2, sx, sy, sz);
	hipDeviceSynchronize();
}


//multiply double complex
extern "C"
void multidcomplex3Dgpu(dComplex *d_odata, dComplex *d_idata1, dComplex *d_idata2, size_t sx, size_t sy, size_t sz){
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sx, blockSize3Dx), iDivUp(sy, blockSize3Dy), iDivUp(sz, blockSize3Dz));
	multidcomplex3Dkernel<<<grids, threads >>>(d_odata, d_idata1, d_idata2, sx, sy, sz);
	hipDeviceSynchronize();
}

//divide
template <class T>
void div3Dgpu(T *d_odata, T *d_idata1, T *d_idata2, size_t sx, size_t sy, size_t sz){
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sx, blockSize3Dx), iDivUp(sy, blockSize3Dy), iDivUp(sz, blockSize3Dz));
	div3Dkernel<T> <<<grids, threads>>>(d_odata, d_idata1, d_idata2, sx, sy, sz);
	hipDeviceSynchronize();
}
template void div3Dgpu<int>(int *d_odata, int *d_idata1, int *d_idata2, size_t sx, size_t sy, size_t sz);
template void div3Dgpu<float>(float *d_odata, float *d_idata1, float *d_idata2, size_t sx, size_t sy, size_t sz);
template void div3Dgpu<double>(double *d_odata, double *d_idata1, double *d_idata2, size_t sx, size_t sy, size_t sz);

//conjugation of complex
extern "C"
void conj3Dgpu(fComplex *d_odata, fComplex *d_idata, size_t sx, size_t sy, size_t sz){
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sx, blockSize3Dx), iDivUp(sy, blockSize3Dy), iDivUp(sz, blockSize3Dz));
	conj3Dkernel <<<grids, threads>>>(d_odata, d_idata, sx, sy, sz);
	hipDeviceSynchronize();
}

// sumarization
// sumgpu 1: small data size
template <class T>
T sumgpu(T *d_idata, int totalSize){
	int gridSize = iDivUp(totalSize, blockSize);
	bool nIsPow2 = isPow2(totalSize);
	int smemSize = (blockSize <= 32) ? 2 * blockSize * sizeof(T) : blockSize * sizeof(T);
	T *h_temp = NULL, *d_temp = NULL;
	h_temp = (T *)malloc(gridSize * sizeof(T));
	hipMalloc((void **)&d_temp, gridSize * sizeof(T));

	sumgpukernel<T><<<gridSize, blockSize, smemSize>>>(
		d_idata,
		d_temp,
		totalSize,
		nIsPow2
		);
	hipDeviceSynchronize();
	hipMemcpy(h_temp, d_temp, gridSize * sizeof(T), hipMemcpyDeviceToHost);
	T sumValue = 0;
	for (int i = 0; i < gridSize; i++){
		sumValue += h_temp[i];
	}
	free(h_temp);
	hipFree(d_temp);
	return sumValue;
}

template int sumgpu<int>(int *d_idata,  int totalSize);
template float sumgpu<float>(float *d_idata,  int totalSize);
template double sumgpu<double>(double *d_idata,  int totalSize);

// sumgpu 2: huge data size (3D data)
template <class T>
double sum3Dgpu(T *d_idata, size_t sx, size_t sy, size_t sz){
	size_t sxy = sx * sy;
	double *h_temp = NULL, *d_temp = NULL;
	h_temp = (double *)malloc(sxy * sizeof(double));
	hipMalloc((void **)&d_temp, sxy * sizeof(double));
	dim3 threads(blockSize2Dx, blockSize2Dy, 1);
	dim3 grids(iDivUp(sx, threads.x), iDivUp(sy, threads.y));
	reduceZ<T> <<<grids, threads >>>(d_idata, d_temp, sx, sy, sz); 
	hipDeviceSynchronize();
	hipMemcpy(h_temp, d_temp, sxy * sizeof(double), hipMemcpyDeviceToHost); 
	double sumValue = 0; 
	for (size_t i = 0; i < sxy; i++)
		sumValue += h_temp[i];
	free(h_temp); 
	hipFree(d_temp);
	return sumValue;
}

template double sum3Dgpu<int>(int *d_idata,  size_t sx, size_t sy, size_t sz);
template double sum3Dgpu<float>(float *d_idata,  size_t sx, size_t sy, size_t sz);
template double sum3Dgpu<double>(double *d_idata,  size_t sx, size_t sy, size_t sz);

// sumgpu 3: small data (1D data)
template <class T>
T sumgpu1D(T *d_idata,  size_t totalSize){
	const size_t r = 5; // a rough number, need further optimization
	size_t tempSize = r * blockSize;
	T *h_temp = NULL, *d_temp = NULL;
	h_temp = (T *)malloc(tempSize * sizeof(T));
	hipMalloc((void **)&d_temp, tempSize * sizeof(T));
	sumgpu1Dkernel<T> <<<r, blockSize >>>(
		d_idata,
		d_temp,
		totalSize
		);
	hipDeviceSynchronize();
	hipMemcpy(h_temp, d_temp, tempSize * sizeof(T), hipMemcpyDeviceToHost);
	T sumValue = 0;
	for (int i = 0; i < tempSize; i++){
		sumValue += h_temp[i];
	}
	free(h_temp);
	hipFree(d_temp);
	return sumValue;
}
template int sumgpu1D<int>(int *d_idata,  size_t totalSize);
template float sumgpu1D<float>(float *d_idata,  size_t totalSize);
template double sumgpu1D<double>(double *d_idata,  size_t totalSize);

// max3Dgpu: find max value and coordinates
template <class T>
T max3Dgpu(size_t *corXYZ, T *d_idata, size_t sx, size_t sy, size_t sz){
	size_t sx0 = 0, sy0 = 0, sz0 = 0;
	T *d_temp1 = NULL, *h_temp1 = NULL;
	size_t *d_temp2 = NULL, *h_temp2 = NULL;
	hipMalloc((void **)&d_temp1, sx*sy *sizeof(T));
	hipMalloc((void **)&d_temp2, sx*sy *sizeof(size_t));
	h_temp1 = (T *)malloc(sx*sy * sizeof(T));
	h_temp2 = (size_t *)malloc(sx*sy * sizeof(size_t));
	dim3 threads(blockSize2Dx, blockSize2Dy, 1);
	dim3 grids(iDivUp(sx, threads.x), iDivUp(sy, threads.y));
	maxZkernel<T> <<<grids, threads >>>(d_idata, d_temp1, d_temp2, sx, sy, sz);
	hipDeviceSynchronize();
	hipMemcpy(h_temp1, d_temp1, sx*sy * sizeof(T), hipMemcpyDeviceToHost);
	hipMemcpy(h_temp2, d_temp2, sx*sy * sizeof(size_t), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	T peakValue = h_temp1[0];
	T t;
	for (size_t i = 0; i < sx; i++){
		for (size_t j = 0; j < sy; j++){
			t = h_temp1[i + j * sx];
			if (peakValue < t){
				peakValue = t;
				sx0 = i; 
				sy0 = j;
				sz0 = h_temp2[i + j * sx];
			}
		}
	}
	corXYZ[0] = sx0; corXYZ[1] = sy0; corXYZ[2] = sz0;
	free(h_temp1); free(h_temp2);
	hipFree(d_temp1); hipFree(d_temp2);
	return peakValue;
}
template int max3Dgpu<int>(size_t *corXYZ, int *d_idata, size_t sx, size_t sy, size_t sz);
template float max3Dgpu<float>(size_t *corXYZ, float *d_idata, size_t sx, size_t sy, size_t sz);
template double max3Dgpu<double>(size_t *corXYZ, double *d_idata, size_t sx, size_t sy, size_t sz);

// max with a single value
template <class T>
void maxvalue3Dgpu(T *d_odata, T *d_idata1, T d_idata2, size_t sx, size_t sy, size_t sz){
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sx, blockSize3Dx), iDivUp(sy, blockSize3Dy), iDivUp(sz, blockSize3Dz));
	maxvalue3Dgpukernel<T><<<grids, threads >>>(d_odata, d_idata1, d_idata2, sx, sy, sz);
	hipDeviceSynchronize();
}
template void maxvalue3Dgpu<int>(int *d_odata, int *d_idata1, int d_idata2, size_t sx, size_t sy, size_t sz);
template void maxvalue3Dgpu<float>(float *d_odata, float *d_idata1, float d_idata2, size_t sx, size_t sy, size_t sz);
template void maxvalue3Dgpu<double>(double *d_odata, double *d_idata1, double d_idata2, size_t sx, size_t sy, size_t sz);


// maximum projection
template <class T>
void maxprojection(T *d_odata, T *d_idata, size_t sx, size_t sy, size_t sz, int pDirection){
	size_t psx, psy, psz;
	if (pDirection == 1){
		psx = sx; psy = sy; psz = sz;
	}
	if (pDirection == 2){
		psx = sz; psy = sx; psz = sy;
	}
	if (pDirection == 3){
		psx = sy; psy = sz; psz = sx;
	}
	dim3 threads(blockSize2Dx, blockSize2Dy, 1);
	dim3 grids(iDivUp(psx, threads.x), iDivUp(psy, threads.y));
	maxprojectionkernel<T> <<<grids, threads >>>(d_odata, d_idata, sx, sy, sz, psx, psy, psz, pDirection);
	hipDeviceSynchronize();
}

template void maxprojection<int>(int *d_odata, int *d_idata, size_t sx, size_t sy, size_t sz, int pDirection);
template void maxprojection<float>(float *d_odata, float *d_idata, size_t sx, size_t sy, size_t sz, int pDirection);
template void maxprojection<double>(double *d_odata, double *d_idata, size_t sx, size_t sy, size_t sz, int pDirection);
//Other functions
template <class T>
void changestorageordergpu(T *d_odata, T *d_idata, size_t sx, size_t sy, size_t sz, int orderMode){
	//orderMode
	// 1: change tiff storage order to C storage order
	//-1: change C storage order to tiff storage order
	assert(d_odata != d_idata);
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sx, blockSize3Dx), iDivUp(sy, blockSize3Dy), iDivUp(sz, blockSize3Dz));
	changestorageordergpukernel<T><<<grids, threads>>>(d_odata, d_idata, sx, sy, sz, orderMode);
	hipDeviceSynchronize();
}
template void changestorageordergpu<int>(int *d_odata, int *d_idata, size_t sx, size_t sy, size_t sz, int orderMode);
template void changestorageordergpu<float>(float *d_odata, float *d_idata, size_t sx, size_t sy, size_t sz, int orderMode);
template void changestorageordergpu<double>(double *d_odata, double *d_idata, size_t sx, size_t sy, size_t sz, int orderMode);

// rotate 90/-90 degree by axis
template <class T>
void rotbyyaxis(T *d_odata, T *d_idata, size_t sx, size_t sy, size_t sz, int rotDirection){
	//rot direction
	// 1: rotate 90 deg around Y axis
	//-1: rotate -90 deg around Y axis
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sx, blockSize3Dx), iDivUp(sy, blockSize3Dy), iDivUp(sz, blockSize3Dz));
	rotbyyaxiskernel<T> <<<grids, threads >>>(d_odata, d_idata, sx, sy, sz, rotDirection);
	hipDeviceSynchronize();
}
template void rotbyyaxis<int>(int *d_odata, int *d_idata, size_t sx, size_t sy, size_t sz, int rotDirection);
template void rotbyyaxis<float>(float *d_odata, float *d_idata, size_t sx, size_t sy, size_t sz, int rotDirection);
template void rotbyyaxis<double>(double *d_odata, double *d_idata, size_t sx, size_t sy, size_t sz, int rotDirection);

/*
// rotate any degree by y axis: matrix for affine transformation
void rot3Dbyyaxis(float *d_odata, float theta, int sx, int sz, int sx2, int sz2){
// Rotation matrix:translation (-sx2/2, -sz2/2) --> rotation--> translation back(sx/2,sy/2)
//	1	0	0	sx / 2			cos(theta)	0	sin(theta)	0		1	0	0	-sx2/2
//	0	1	0		0		*		0		1		0		0	*	0	1	0	0	
//	0	0	1	sz / 2			-sin(theta)	0	cos(theta)	0		0	0	1	-sz2/2
//	0	0	0		1				0		0		0		1		0	0	0	1
	d_odata[0] = cos(theta); d_odata[1] = 0; d_odata[2] = sin(theta);
	d_odata[3] = sx / 2 - sx2 / 2 * cos(theta) - sz2 / 2 * sin(theta);
	d_odata[4] = 0; d_odata[5] = 1; d_odata[6] = 0; d_odata[7] = 0;
	d_odata[8] = -sin(theta); d_odata[9] = 0; d_odata[10] = cos(theta);
	d_odata[11] = sz / 2 + sx2 / 2 * sin(theta) - sz2 / 2 * cos(theta);
}
*/

void p2matrix(float *m, float *x){

	m[0] = x[4], m[1] = x[5], m[2] = x[6], m[3] = x[1];
	m[4] = x[7], m[5] = x[8], m[6] = x[9], m[7] = x[2];
	m[8] = x[10], m[9] = x[11], m[10] = x[12], m[11] = x[3];

	/*
	m[0] = x[1], m[1] = x[2], m[2] = x[3], m[3] = x[4];
	m[4] = x[5], m[5] = x[6], m[6] = x[7], m[7] = x[8];
	m[8] = x[9], m[9] = x[10], m[10] = x[11], m[11] = x[12];
	*/
}
void matrix2p(float *m, float *x){
	x[0] = 0;

	x[1] = m[3], x[2] = m[7], x[3] = m[11], x[4] = m[0];
	x[5] = m[1], x[6] = m[2], x[7] = m[4], x[8] = m[5];
	x[9] = m[6], x[10] = m[8], x[11] = m[9], x[12] = m[10];

	/*
	x[1] = m[0], x[2] = m[1], x[3] = m[2], x[4] = m[3];
	x[5] = m[4], x[6] = m[5], x[7] = m[6], x[8] = m[7];
	x[9] = m[8], x[10] = m[9], x[11] = m[10], x[12] = m[11];
	*/
}


extern "C" void matrixmultiply(float * m, float *m1, float *m2){//for transformation matrix calcution only
	m[0] = m1[0] * m2[0] + m1[1] * m2[4] + m1[2] * m2[8];
	m[1] = m1[0] * m2[1] + m1[1] * m2[5] + m1[2] * m2[9];
	m[2] = m1[0] * m2[2] + m1[1] * m2[6] + m1[2] * m2[10];
	m[3] = m1[0] * m2[3] + m1[1] * m2[7] + m1[2] * m2[11] + m1[3];

	m[4] = m1[4] * m2[0] + m1[5] * m2[4] + m1[6] * m2[8];
	m[5] = m1[4] * m2[1] + m1[5] * m2[5] + m1[6] * m2[9];
	m[6] = m1[4] * m2[2] + m1[5] * m2[6] + m1[6] * m2[10];
	m[7] = m1[4] * m2[3] + m1[5] * m2[7] + m1[6] * m2[11] + m1[7];

	m[8] = m1[8] * m2[0] + m1[9] * m2[4] + m1[10] * m2[8];
	m[9] = m1[8] * m2[1] + m1[9] * m2[5] + m1[10] * m2[9];
	m[10] = m1[8] * m2[2] + m1[9] * m2[6] + m1[10] * m2[10];
	m[11] = m1[8] * m2[3] + m1[9] * m2[7] + m1[10] * m2[11] + m1[11];
	//**** 12 13 14 15 never change ****
	//no need to calculate m[12,13,14,15]:0 0 0 1

	/*
	m[0] = m1[0] * m2[0] + m1[1] * m2[4] + m1[2] * m2[8] + m1[3] * m2[12];
	m[1] = m1[0] * m2[1] + m1[1] * m2[5] + m1[2] * m2[9] + m1[3] * m2[13];
	m[2] = m1[0] * m2[2] + m1[1] * m2[6] + m1[2] * m2[10] + m1[3] * m2[14];
	m[3] = m1[0] * m2[3] + m1[1] * m2[7] + m1[2] * m2[11] + m1[3] * m2[15];

	m[4] = m1[4] * m2[0] + m1[5] * m2[4] + m1[6] * m2[8] + m1[7] * m2[12];
	m[5] = m1[4] * m2[1] + m1[5] * m2[5] + m1[6] * m2[9] + m1[7] * m2[13];
	m[6] = m1[4] * m2[2] + m1[5] * m2[6] + m1[6] * m2[10] + m1[7] * m2[14];
	m[7] = m1[4] * m2[3] + m1[5] * m2[7] + m1[6] * m2[11] + m1[7] * m2[15];

	m[8] = m1[8] * m2[0] + m1[9] * m2[4] + m1[10] * m2[8] + m1[11] * m2[12];
	m[9] = m1[8] * m2[1] + m1[9] * m2[5] + m1[10] * m2[9] + m1[11] * m2[13];
	m[10] = m1[8] * m2[2] + m1[9] * m2[6] + m1[10] * m2[10] + m1[11] * m2[14];
	m[11] = m1[8] * m2[3] + m1[9] * m2[7] + m1[10] * m2[11] + m1[11] * m2[15];

	m[12] = m1[12] * m2[0] + m1[13] * m2[4] + m1[14] * m2[8] + m1[15] * m2[12];
	m[13] = m1[12] * m2[1] + m1[13] * m2[5] + m1[14] * m2[9] + m1[15] * m2[13];
	m[14] = m1[12] * m2[2] + m1[13] * m2[6] + m1[14] * m2[10] + m1[15] * m2[14];
	m[15] = m1[12] * m2[3] + m1[13] * m2[7] + m1[14] * m2[11] + m1[15] * m2[15];
	*/
}


extern "C" void rot2matrix(float * p_out, float theta, long long int sx, long long int sy, long long int sz, int rotAxis){
	//p_out: 12 elements
	//theta: rotation angle
	//sx, sy, sz: images size
	////rotAxis
	// 1: rotate theta around X axis
	// 2: rotate theta around Y axis
	// 3: rotate theta around Z axis

	long long int sNew;
	float *p_temp, *p_temp1, *p_temp2, *p_temp3;
	p_temp = (float *)malloc(16 * sizeof(float));
	p_temp1 = (float *)malloc(16 * sizeof(float));
	p_temp2 = (float *)malloc(16 * sizeof(float));
	p_temp3 = (float *)malloc(16 * sizeof(float));
	for (int i = 0; i < 15; i++){
		p_temp[i] = p_temp1[i] = p_temp2[i] = p_temp3[i] = 0;
	}
	p_temp[15] = p_temp1[15] = p_temp2[15] = p_temp3[15] = 1; //**** 12 13 14 15 never change ****

	// matrix: p_temp1 * p_temp2 * p_temp3

	if (rotAxis == 1){//Rotate by x axis
		// Rotation matrix:translation (0, -sx2/2, -sz2/2) --> rotation--> translation back(0,sy/2,sz/2)
		//	1	0	0		0			1		0			0		0		1	0	0	0
		//	0	1	0	sx / 2		*	0	cos(theta)	sin(theta)	0	*	0	1	0	-sy2/2	
		//	0	0	1	sz / 2			0	-sin(theta)	cos(theta)	0		0	0	1	-sz2/2
		//	0	0	0		1			0		0			0		1		0	0	0	1
		p_temp1[0] = p_temp1[5] = p_temp1[10] = 1;
		p_temp1[7] = sy / 2; p_temp1[11] = sz / 2;

		p_temp2[0] = 1; p_temp2[1] = 0; p_temp2[2] = 0; p_temp2[3] = 0;
		p_temp2[4] = 0; p_temp2[5] = cos(theta); p_temp2[6] = sin(theta); p_temp2[7] = 0;
		p_temp2[8] = 0; p_temp2[9] = -sin(theta); p_temp2[10] = cos(theta); p_temp2[11] = 0;

		sNew = round(sqrt(sy * sy + sz*sz));
		p_temp3[0] = p_temp3[5] = p_temp3[10] = 1;
		p_temp3[7] = - sNew / 2; p_temp3[11] = - sNew / 2; 
	}

	if (rotAxis == 2){//Rotate by y axis

		// Rotation matrix:translation (-sx2/2, 0, -sz2/2) --> rotation--> translation back(sx/2,0,sz/2)
		//	1	0	0	sx / 2			cos(theta)	0	-sin(theta)	0		1	0	0	-sx2/2
		//	0	1	0		0		*		0		1		0		0	*	0	1	0	0	
		//	0	0	1	sz / 2			sin(theta)	0	cos(theta)	0		0	0	1	-sz2/2
		//	0	0	0		1				0		0		0		1		0	0	0	1

		p_temp1[0] = p_temp1[5] = p_temp1[10] = 1;
		p_temp1[3] = sx / 2; p_temp1[11] = sz / 2;

		p_temp2[0] = cos(theta); p_temp2[1] = 0; p_temp2[2] = -sin(theta); p_temp2[3] = 0;
		p_temp2[4] = 0; p_temp2[5] = 1; p_temp2[6] = 0; p_temp2[7] = 0;
		p_temp2[8] = sin(theta); p_temp2[9] = 0; p_temp2[10] = cos(theta); p_temp2[11] = 0;

		sNew = round(sqrt(sx * sx + sz*sz));
		p_temp3[0] = p_temp3[5] = p_temp3[10] = 1;
		p_temp3[3] = -sNew / 2; p_temp3[11] = -sNew / 2;
	}

	if (rotAxis == 3){//Rotate by z axis
		// Rotation matrix:translation (-sx2/2,-sy2/2, 0) --> rotation--> translation back(sx/2,sy/2,0)
		//	1	0	0	sx / 2			cos(theta)	sin(theta)	0	0		1	0	0	-sx2/2
		//	0	1	0	sy / 2		*	-sin(theta)	cos(theta)	0	0	*	0	1	0	-sy2/2	
		//	0	0	1		0				0			0		1	0		0	0	1	0
		//	0	0	0		1				0			0		0	1		0	0	0	1

		p_temp1[0] = p_temp1[5] = p_temp1[10] = 1;
		p_temp1[3] = sx / 2; p_temp1[7] = sy / 2;

		p_temp2[0] = cos(theta); p_temp2[1] = sin(theta); p_temp2[2] = 0; p_temp2[3] = 0;
		p_temp2[4] = -sin(theta); p_temp2[5] = cos(theta); p_temp2[6] = 0; p_temp2[7] = 0;
		p_temp2[8] = 0; p_temp2[9] = 0; p_temp2[10] = 1; p_temp2[11] = 0;

		sNew = round(sqrt(sx * sx + sy*sy));
		p_temp3[0] = p_temp3[5] = p_temp3[10] = 1;
		p_temp3[3] = -sNew / 2; p_temp3[7] = -sNew / 2;
	}


	matrixmultiply(p_temp, p_temp1, p_temp2);
	matrixmultiply(p_out, p_temp, p_temp3);

	free(p_temp);
	free(p_temp1);
	free(p_temp2);
	free(p_temp3);
}

extern "C" void dof9tomatrix(float * p_out, float *p_dof, int dofNum){
	//p_out: 12 elements
	//p_dof: 10 elements: 0 x y z alpha beta theda a b c 
	//dofNum: 3, 6, 7 or 9
	float *p_temp1, *p_temp2, *p_temp3;
	p_temp1 = (float *)malloc(16 * sizeof(float));
	p_temp2 = (float *)malloc(16 * sizeof(float));
	p_temp3 = (float *)malloc(16 * sizeof(float));
	for (int i = 0; i < 15; i++){
		p_temp1[i] = p_temp2[i] = p_temp3[i] = 0;
	}
	p_temp1[15] = p_temp2[15] = p_temp3[15] = 1; //**** 12 13 14 15 never change ****

	float x, y, z, alpha, beta, theta, a, b, c;
	if (dofNum == 3){//translation
		x = p_dof[1];
		y = p_dof[2];
		z = p_dof[3];
		alpha = 0;
		beta = 0;
		theta = 0;
		a = 1;
		b = 1;
		c = 1;
	}
	else if (dofNum == 6){//rigid body: translation, rotation
		x = p_dof[1];
		y = p_dof[2];
		z = p_dof[3];
		alpha = p_dof[4] / 57.3;
		beta = p_dof[5] / 57.3;
		theta = p_dof[6] / 57.3;
		a = 1;
		b = 1;
		c = 1;
	}
	else if (dofNum == 7){//translation,rotation, scale equelly in 3 dimemsions 
		x = p_dof[1];
		y = p_dof[2];
		z = p_dof[3];
		alpha = p_dof[4] / 57.3;
		beta = p_dof[5] / 57.3;
		theta = p_dof[6] / 57.3;
		a = p_dof[7];
		b = p_dof[7];
		c = p_dof[7];
	}
	else if (dofNum == 9){//translation,rotation,scale
		x = p_dof[1];
		y = p_dof[2];
		z = p_dof[3];
		alpha = p_dof[4] / 57.3;
		beta = p_dof[5] / 57.3;
		theta = p_dof[6] / 57.3;
		a = p_dof[7];
		b = p_dof[8];
		c = p_dof[9];
	}

	//translation
	// 1	0	0	x
	// 0	1	0	y
	// 0	0	1	z
	// 0	0	0	1
	p_temp2[3] = x;
	p_temp2[7] = y;
	p_temp2[11] = z;
	// scaling
	// a	0	0	0
	// 0	b	0	0
	// 0	0	c	0
	// 0	0	0	1
	p_temp2[0] = a;
	p_temp2[5] = b;
	p_temp2[10] = c;
	// rotating by Z axis
	// cos(alpha)	sin(alpha)	0	0
	// -sin(alpha)	cos(alpha)	0	0
	// 0			0			1	0
	// 0			0			0	1
	p_temp3[0] = cos(alpha); p_temp3[1] = sin(alpha); p_temp3[2] = 0; p_temp3[3] = 0;
	p_temp3[4] = -sin(alpha); p_temp3[5] = cos(alpha); p_temp3[6] = 0; p_temp3[7] = 0;
	p_temp3[8] = 0; p_temp3[9] = 0; p_temp3[10] = 1; p_temp3[11] = 0;
	//p_temp3[15] = 1;
	matrixmultiply(p_temp1, p_temp2, p_temp3);
	// rotating by X axis
	// 1	0			0			0
	// 0	cos(beta)	sin(beta)	0
	// 0	-sin(beta)	cos(beta)	0
	// 0	0			0			1
	p_temp3[0] = 1; p_temp3[1] = 0; p_temp3[2] = 0; p_temp3[3] = 0;
	p_temp3[4] = 0; p_temp3[5] = cos(beta); p_temp3[6] = sin(beta); p_temp3[7] = 0;
	p_temp3[8] = 0; p_temp3[9] = -sin(beta); p_temp3[10] = cos(beta); p_temp3[11] = 0;
	//p_temp3[15] = 1;
	matrixmultiply(p_temp2, p_temp1, p_temp3);
	// rotating by Y axis
	// cos(theta)	0	-sin(theta)		0
	// 0			1	0				0
	// sin(theta)	0	cos(theta)		0
	// 0			0	0				1
	p_temp3[0] = cos(theta); p_temp3[1] = 0; p_temp3[2] = -sin(theta); p_temp3[3] = 0;
	p_temp3[4] = 0; p_temp3[5] = 1; p_temp3[6] = 0; p_temp3[7] = 0;
	p_temp3[8] = sin(theta); p_temp3[9] = 0; p_temp3[10] = cos(theta); p_temp3[11] = 0;
	//p_temp3[15] = 1;
	matrixmultiply(p_out, p_temp2, p_temp3);

	free(p_temp1);
	free(p_temp2);
	free(p_temp3);
}

template <class T>
void circshiftgpu(T *d_odata, T *d_idata, long long int sx, long long int sy, long long int sz, long long int dx, long long int dy, long long int dz){
	assert(d_odata != d_idata);
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sx, blockSize3Dx), iDivUp(sy, blockSize3Dy), iDivUp(sz, blockSize3Dz));
	circshiftgpukernel<T> <<<grids, threads >>>(d_odata, d_idata, sx, sy, sz, dx, dy, dz);
	hipDeviceSynchronize();
}
template void circshiftgpu<int>(int *d_odata, int *d_idata, long long int sx, long long int sy, long long int sz, long long int dx, long long int dy, long long int dz);
template void circshiftgpu<float>(float *d_odata, float *d_idata, long long int sx, long long int sy, long long int sz, long long int dx, long long int dy, long long int dz);
template void circshiftgpu<double>(double *d_odata, double *d_idata, long long int sx, long long int sy, long long int sz, long long int dx, long long int dy, long long int dz);
template <class T>
void imshiftgpu(T *d_odata, T *d_idata, long long int sx, long long int sy, long long int sz, long long int dx, long long int dy, long long int dz) {
	assert(d_odata != d_idata);
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sx, blockSize3Dx), iDivUp(sy, blockSize3Dy), iDivUp(sz, blockSize3Dz));
	imshiftgpukernel<T> << <grids, threads >> >(d_odata, d_idata, sx, sy, sz, dx, dy, dz);
	hipDeviceSynchronize();
}
template void imshiftgpu<int>(int *d_odata, int *d_idata, long long int sx, long long int sy, long long int sz, long long int dx, long long int dy, long long int dz);
template void imshiftgpu<float>(float *d_odata, float *d_idata, long long int sx, long long int sy, long long int sz, long long int dx, long long int dy, long long int dz);
template void imshiftgpu<double>(double *d_odata, double *d_idata, long long int sx, long long int sy, long long int sz, long long int dx, long long int dy, long long int dz);

extern "C" void CopyTranMatrix(float *x, int dataSize){
	hipMemcpyToSymbol(HIP_SYMBOL(d_aff), x, dataSize, 0, hipMemcpyHostToDevice);
}


template <class T>
void cudacopyhosttoarray(hipArray *d_Array, hipChannelFormatDesc channelDesc, T *h_idata, size_t sx, size_t sy, size_t sz){
	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr((void*)h_idata, sx*sizeof(T), sx, sy);
	copyParams.dstArray = d_Array;
	copyParams.extent = make_hipExtent(sx, sy, sz);
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);
	hipDeviceSynchronize();
}
template void
cudacopyhosttoarray<unsigned short>(hipArray *d_Array, hipChannelFormatDesc channelDesc, unsigned short *h_idata, size_t sx, size_t sy, size_t sz);
template void
cudacopyhosttoarray<float>(hipArray *d_Array, hipChannelFormatDesc channelDesc, float *h_idata, size_t sx, size_t sy, size_t sz);

template <class T>
void cudacopydevicetoarray(hipArray *d_Array, hipChannelFormatDesc channelDesc, T *d_idata, size_t sx, size_t sy, size_t sz){
	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr((void*)d_idata, sx*sizeof(T), sx, sy);
	copyParams.dstArray = d_Array;
	copyParams.extent = make_hipExtent(sx, sy, sz);
	copyParams.kind = hipMemcpyDeviceToDevice;
	hipMemcpy3D(&copyParams);
	hipDeviceSynchronize();
}
template void
cudacopydevicetoarray<unsigned short>(hipArray *d_Array, hipChannelFormatDesc channelDesc, unsigned short *d_idata, size_t sx, size_t sy, size_t sz);
template void
cudacopydevicetoarray<float>(hipArray *d_Array, hipChannelFormatDesc channelDesc, float *d_idata, size_t sx, size_t sy, size_t sz);


extern "C" void BindTexture(hipArray *d_Array, hipChannelFormatDesc channelDesc){
	// set texture parameters
	tex.addressMode[0] = hipAddressModeWrap;
	tex.addressMode[1] = hipAddressModeWrap;
	tex.addressMode[2] = hipAddressModeWrap;
	tex.filterMode = hipFilterModeLinear;
	tex.normalized = false; //NB coordinates in [0,1]
	// Bind the array to the texture
	hipBindTextureToArray(tex, d_Array, channelDesc);
	hipDeviceSynchronize();
}

extern "C" void BindTexture2(hipArray *d_Array, hipChannelFormatDesc channelDesc) {
	// set texture parameters
	tex.addressMode[0] = hipAddressModeWrap;
	tex.addressMode[1] = hipAddressModeWrap;
	tex.addressMode[2] = hipAddressModeWrap;
	tex.filterMode = hipFilterModeLinear;
	tex.normalized = false; //NB coordinates in [0,1]
							// Bind the array to the texture
	hipBindTextureToArray(tex2, d_Array, channelDesc);
	hipDeviceSynchronize();
}

extern "C" void BindTexture16(hipArray *d_Array, hipChannelFormatDesc channelDesc){
	// set texture parameters
	tex.addressMode[0] = hipAddressModeWrap;
	tex.addressMode[1] = hipAddressModeWrap;
	tex.addressMode[2] = hipAddressModeWrap;
	tex.filterMode = hipFilterModeLinear;
	tex.normalized = false; //NB coordinates in [0,1]
	// Bind the array to the texture
	hipBindTextureToArray(tex16, d_Array, channelDesc);
	hipDeviceSynchronize();
}

extern "C" void UnbindTexture(){
	hipUnbindTexture(tex);
	hipDeviceSynchronize();
}

extern "C" void UnbindTexture2() {
	hipUnbindTexture(tex2);
	hipDeviceSynchronize();
}

extern "C" void UnbindTexture16(){
	hipUnbindTexture(tex16);
	hipDeviceSynchronize();
}

extern "C" void AccessTexture(float x, float y,float z){
	dim3 threads(2, 2, 2);
	accesstexturekernel <<<1, threads >>>(x, y, z);
	hipDeviceSynchronize();
}

template <class T> 
void affineTransform(T *d_s, long long int sx, long long int sy, long long int sz, long long int sx2, long long int sy2, long long int sz2){
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grid(iDivUp(sx, threads.x), iDivUp(sy, threads.y), iDivUp(sz, threads.z));
	affinetransformkernel<T><<<grid, threads >>>(d_s, sx, sy, sz, sx2, sy2, sz2);
	hipDeviceSynchronize();
}
template void
affineTransform<unsigned short>(unsigned short *d_s, long long int sx, long long int sy, long long int sz, long long int sx2, long long int sy2, long long int sz2);
template void 
affineTransform<float>(float *d_s, long long int sx, long long int sy, long long int sz, long long int sx2, long long int sy2, long long int sz2);

float corrfunc(float *d_t, float sd_t, float *aff, long long int sx, 
	long long int sy, long long int sz, long long int sx2, long long int sy2, long long int sz2){
	// temp bufs
	long long int sxy = sx * sy;
	double *d_temp1 = NULL, *d_temp2 = NULL;
	hipMalloc((void **)&d_temp1, sxy * sizeof(double));
	hipMalloc((void **)&d_temp2, sxy * sizeof(double));
	//copy aff to GPU const
	hipMemcpyToSymbol(HIP_SYMBOL(d_aff), aff, 12 * sizeof(float), 0, hipMemcpyHostToDevice);// copy host affine matrix to device const
	dim3 threads(blockSize2Dx, blockSize2Dy, 1);
	dim3 grids(iDivUp(sx, threads.x), iDivUp(sy, threads.y));
	corrkernel<<<grids, threads>>>( d_t, // the source image is texture, trans matrix is const
		d_temp1, d_temp2, sx, sy, sz, sx2, sy2, sz2);
	hipDeviceSynchronize();
	double sqrSum = 0, corrSum = 0;
	if (sxy > 100000){ // if count more than 100000, use gpu to perform sum
		sqrSum = sumgpu1D(d_temp1,  sxy);
		corrSum = sumgpu1D(d_temp2, sxy);
	}
	else{
		double *h_temp = NULL;
		h_temp = (double *)malloc(sx*sy * sizeof(double));
		hipMemcpy(h_temp, d_temp1, sxy * sizeof(double), hipMemcpyDeviceToHost);
		for (int i = 0; i < sxy; i++)
			sqrSum += h_temp[i];
		hipMemcpy(h_temp, d_temp2, sxy * sizeof(double), hipMemcpyDeviceToHost);
		for (int i = 0; i < sxy; i++)
			corrSum += h_temp[i];
		free(h_temp);
	}
	hipFree(d_temp1); 
	hipFree(d_temp2); 
	if (sqrt(sqrSum) == 0) return -2.0;
	return (float)(corrSum / sqrt(sqrSum)) / sd_t;
}

extern "C" void BindTexture2D(hipArray *d_Array, hipChannelFormatDesc channelDesc){
	// set texture parameters
	tex2D1.addressMode[0] = hipAddressModeWrap;
	tex2D1.addressMode[1] = hipAddressModeWrap;
	tex2D1.filterMode = hipFilterModeLinear;
	tex2D1.normalized = false;    // access with normalized texture coordinates

	// Bind the array to the texture
	hipBindTextureToArray(tex2D1, d_Array, channelDesc);
}

extern "C" void UnbindTexture2D(
	){
	hipUnbindTexture(tex2D1);
}

extern "C"
void affineTransform2D(float *d_t, int sx, int sy, int sx2, int sy2){
	dim3 threads(blockSize2Dx, blockSize2Dy, 1);
	dim3 grids(iDivUp(sx, threads.x), iDivUp(sy, threads.y));
	affineTransform2Dkernel <<<grids, threads >>>(d_t, sx, sy, sx2, sy2);
	hipDeviceSynchronize();
}

float corrfunc2D(float *d_t, float sd_t, float *aff, long long int sx, long long int sy, long long int sx2, long long int sy2){
	//copy aff to GPU const
	hipMemcpyToSymbol(HIP_SYMBOL(d_aff), aff, 6 * sizeof(float), 0, hipMemcpyHostToDevice);// copy host affine matrix to device const
	long long int totalSize = sx*sy;
	float *d_sqr = NULL, *d_corr = NULL, *h_temp = NULL;
	hipMalloc((void **)&d_sqr, totalSize * sizeof(float));
	hipMalloc((void **)&d_corr, totalSize * sizeof(float));
	h_temp = (float *)malloc(totalSize * sizeof(float));
	dim3 threads(blockSize2Dx, blockSize2Dy, 1);
	dim3 grids(iDivUp(sx, threads.x), iDivUp(sy, threads.y));
	corr2Dkernel <<<grids, threads >>>( // the other image is texture, trans matrix is const
		d_t, d_sqr, d_corr, sx, sy, sx2, sy2);
	hipDeviceSynchronize();
	hipMemcpy(h_temp, d_corr, totalSize * sizeof(float), hipMemcpyDeviceToHost);
	double corrSum = sumcpu(h_temp, totalSize);
	hipMemcpy(h_temp, d_sqr, totalSize * sizeof(float), hipMemcpyDeviceToHost);
	double sqrSum = sumcpu(h_temp, totalSize);
	hipFree(d_sqr);
	hipFree(d_corr);
	free(h_temp);
	if (sqrt(sqrSum) == 0) return -2.0;
	return float(corrSum / sqrt(sqrSum))/sd_t;
}

///// CPU interpolation
float lerp(float x, float x1, float x2, float q00, float q01) {
	return ((x2 - x) / (x2 - x1)) * q01 + ((x - x1) / (x2 - x1)) * q00;
}

float bilerp(float x, float y, float x1, float x2, float y1, float y2, float q11, float q12, float q21, float q22) {
	float r1 = lerp(x, x1, x2, q11, q12);
	float r2 = lerp(x, x1, x2, q21, q22);

	return lerp(y, y1, y2, r1, r2);
}

float trilerp(float x, float y, float z, float x1, float x2, float y1, float y2, float z1, float z2, 
	float q111, float q112, float q121, float q122, float q211, float q212, float q221, float q222) {
	float r1 = bilerp(x, y, x1, x2, y1, y2, q111, q112, q121, q122);
	float r2 = bilerp(x, y, x1, x2, y1, y2, q211, q212, q221, q222);
	return lerp(z, z1, z2, r1, r2);
}

float ilerp(float x, float x1, float x2, float q00, float q01) {
	return (x2 - x) * q00 + (x - x1) * q01;
}

float ibilerp(float x, float y, float x1, float x2, float y1, float y2, float q11, float q12, float q21, float q22) {
	float r1 = ilerp(x, x1, x2, q11, q12);
	float r2 = ilerp(x, x1, x2, q21, q22);

	return ilerp(y, y1, y2, r1, r2);
}

float itrilerp(float x, float y, float z, float x1, float x2, float y1, float y2, float z1, float z2,
	float q111, float q112, float q121, float q122, float q211, float q212, float q221, float q222) {
	float r1 = ibilerp(x, y, x1, x2, y1, y2, q111, q112, q121, q122);
	float r2 = ibilerp(x, y, x1, x2, y1, y2, q211, q212, q221, q222);
	return ilerp(z, z1, z2, r1, r2);
}

float ilerp2(float dx1, float dx2, float q00, float q01) {
	return dx2 * q00 + dx1 * q01;
}

float ibilerp2(float dx1, float dx2, float dy1, float dy2, float q11, float q12, float q21, float q22) {
	float r1 = ilerp2(dx1, dx2, q11, q12);
	float r2 = ilerp2(dx1, dx2, q21, q22);

	return ilerp2(dy1, dy2, r1, r2);
}

float itrilerp2(float dx1, float dx2, float dy1, float dy2, float dz1, float dz2,
	float q111, float q112, float q121, float q122, float q211, float q212, float q221, float q222) {
	float r1 = ibilerp2(dx1, dx2, dy1, dy2, q111, q112, q121, q122);
	float r2 = ibilerp2(dx1, dx2, dy1, dy2, q211, q212, q221, q222);
	return ilerp2(dz1, dz2, r1, r2);
}

//output[sz-k-1][j][i] = input[i][j][k]
//d_odata[(sz - k - 1)*sx*sy + j*sx + i] = d_idata[i*sy*sz + j*sz + k];
double corrfunccpu(float *h_s,
	float *h_t,// source stack
	float *aff,
	int sx,
	int sy,
	int sz,
	int sx2,
	int sy2,
	int sz2
	){
	double sqrSum = 0, corrSum = 0;
	int x1, y1, z1, x2, y2, z2;
	float q1, q2, q3, q4, q5, q6, q7, q8;
	float s, t;
	int sxy = sx*sy, sxy2 = sx2*sy2;
	for (int i = 0; i < sx; i++){
		for (int j = 0; j < sy; j++){
			for (int k = 0; k < sz; k++){
				float ix = (float)i;
				float iy = (float)j;
				float iz = (float)k;
				float tx = aff[0] * ix + aff[1] * iy + aff[2] * iz + aff[3];
				float ty = aff[4] * ix + aff[5] * iy + aff[6] * iz + aff[7];
				float tz = aff[8] * ix + aff[9] * iy + aff[10] * iz + aff[11];
				x1 = floor(tx); y1 = floor(ty); z1 = floor(tz);
				x2 = x1 + 1; y2 = y1 + 1; z2 = z1 + 1;
				if ((x1 >= 0) && (y1 >= 0) && (z1 >= 0) && (x2 < sx2) && (y2 < sy2) && (z2 < sz2)){
					// [k*sy*sx + j*sx + i]
					q1 = h_t[z1*sxy2 + y1*sx2 + x1];
					q2 = h_t[z1*sxy2 + y1*sx2 + x2];
					q3 = h_t[z1*sxy2 + y2*sx2 + x1];
					q4 = h_t[z1*sxy2 + y2*sx2 + x2];
					q5 = h_t[z2*sxy2 + y1*sx2 + x1];
					q6 = h_t[z2*sxy2 + y1*sx2 + x2];
					q7 = h_t[z2*sxy2 + y2*sx2 + x1];
					q8 = h_t[z2*sxy2 + y2*sx2 + x2];
					t = itrilerp(tx, ty, tz, x1, x2, y1, y2, z1, z2, q1, q2, q3, q4, q5, q6, q7, q8);
				}
				else
					t = 0;
				s = h_s[k*sxy + j*sx + i];

				sqrSum += (double)t*t;
				corrSum += (double)s*t;
			}
		}
	}
	return (corrSum / sqrt(sqrSum));
}


double corrfunccpu3(float *h_s,
	float *h_t,// source stack
	float *aff,
	int sx,
	int sy,
	int sz,
	int sx2,
	int sy2,
	int sz2
	){
	const float r0 = aff[0], r1 = aff[1], r2 = aff[2], r3 = aff[3], r4 = aff[4], r5= aff[5],
		r6 = aff[6], r7 = aff[7], r8 = aff[8], r9 = aff[9], r10 = aff[10], r11 = aff[11];

	double sqrSum = 0, corrSum = 0;
	float ix, iy, iz, tx, ty, tz;
	int x1, y1, z1, x2, y2, z2;
	float dx1, dy1, dz1, dx2, dy2, dz2;
	float q1, q2, q3, q4, q5, q6, q7, q8;
	float s, t;
	int syz = sy*sz, syz2 = sy2*sz2, x1syz2, x2syz2, y1sz2, y2sz2;
	for (int i = 0; i < sx; i++){
		ix = (float)i;
		for (int j = 0; j < sy; j++){
			iy = (float)j;
			for (int k = 0; k < sz; k++){
				iz = (float)k;
				
				tx = r0 * ix + r1 * iy + r2 * iz + r3;
				ty = r4 * ix + r5 * iy + r6 * iz + r7;
				tz = r8 * ix + r9 * iy + r10 * iz + r11;
				
				x1 = (int)tx; y1 = (int)ty; z1 = (int)tz;
				x2 = x1 + 1; y2 = y1 + 1; z2 = z1 + 1;

				dx1 = tx - (float)x1; dy1 = ty - (float)y1; dz1 = tz - (float)z1;
				dx2 = 1 - dx1; dy2 = 1 - dy1; dz2 = 1 - dz1;
				if (x1 >= 0 && y1 >= 0 && z1 >= 0 && x2 < sx2 && y2 < sy2 && z2 < sz2){
					// [i*sy*sz + j*sz + k]
					x1syz2 = x1*syz2;
					x2syz2 = x2*syz2;
					y1sz2 = y1*sz2;
					y2sz2 = y2*sz2;

					q1 = h_t[x1syz2 + y1sz2 + z1];
					q2 = h_t[x2syz2 + y1sz2 + z1];
					q3 = h_t[x1syz2 + y2sz2 + z1];
					q4 = h_t[x2syz2 + y2sz2 + z1];
					q5 = h_t[x1syz2 + y1sz2 + z2];
					q6 = h_t[x2syz2 + y1sz2 + z2];
					q7 = h_t[x1syz2 + y2sz2 + z2];
					q8 = h_t[x2syz2 + y2sz2 + z2];
					//t = itrilerp2(dx1, dx2, dy1, dy2, dz1, dz2, q1, q2, q3, q4, q5, q6, q7, q8);
					//t = itrilerp(tx, ty, tz, x1, x2, y1, y2, z1, z2, q1, q2, q3, q4, q5, q6, q7, q8);
					t = dz2*(dy2*dx2*q1 + dy2*dx1*q2 + dy1*dx2*q3 + dy1*dx1*q4) + dz1*(dy2*dx2*q5 + dy2*dx1*q6 + dy1*dx2*q7 + dy1*dx1*q8);
					//t = 1;

				}
				else
					t = 0;
				s = h_s[i*syz + j*sz + k];

				sqrSum += (double)t*t;
				corrSum += (double)s*t;
			}
		}
	}
	return (corrSum / sqrt(sqrSum));
}
double corrfunccpu2_old(float *h_s,
	float *h_t,// source stack
	float *aff,
	int sx,
	int sy,
	int sz,
	int sx2,
	int sy2,
	int sz2
	){
	const float r0 = aff[0], r1 = aff[1], r2 = aff[2], r3 = aff[3], r4 = aff[4], r5 = aff[5],
		r6 = aff[6], r7 = aff[7], r8 = aff[8], r9 = aff[9], r10 = aff[10], r11 = aff[11];

	double sqrSum = 0, corrSum = 0;
	float ix, iy, iz, tx, ty, tz;
	int x1, y1, z1, x2, y2, z2;
	float dx1, dy1, dz1, dx2, dy2, dz2;
	float q1, q2, q3, q4, q5, q6, q7, q8;
	float s, t;
	int sxy = sx*sy, sxy2 = sx2*sy2, z1sxy2, z2sxy2, y1sx2, y2sx2;
	for (int i = 0; i < sx; i++){
		ix = (float)i;
		for (int j = 0; j < sy; j++){
			iy = (float)j;
			for (int k = 0; k < sz; k++){
				iz = (float)k;

				tx = r0 * ix + r1 * iy + r2 * iz + r3;
				ty = r4 * ix + r5 * iy + r6 * iz + r7;
				tz = r8 * ix + r9 * iy + r10 * iz + r11;

				x1 = (int)tx; y1 = (int)ty; z1 = (int)tz;
				x2 = x1 + 1; y2 = y1 + 1; z2 = z1 + 1;

				dx1 = tx - (float)x1; dy1 = ty - (float)y1; dz1 = tz - (float)z1;
				dx2 = 1 - dx1; dy2 = 1 - dy1; dz2 = 1 - dz1;
				if (x1 >= 0 && y1 >= 0 && z1 >= 0 && x2 < sx2 && y2 < sy2 && z2 < sz2){
					// [i*sy*sz + j*sz + k]
					z1sxy2 = z1*sxy2;
					z2sxy2 = z2*sxy2;
					y1sx2 = y1*sx2;
					y2sx2 = y2*sx2;

					q1 = h_t[z1sxy2 + y1sx2 + x1];
					q2 = h_t[z1sxy2 + y1sx2 + x2];
					q3 = h_t[z1sxy2 + y2sx2 + x1];
					q4 = h_t[z1sxy2 + y2sx2 + x2];
					q5 = h_t[z2sxy2 + y1sx2 + x1];
					q6 = h_t[z2sxy2 + y1sx2 + x2];
					q7 = h_t[z2sxy2 + y2sx2 + x1];
					q8 = h_t[z2sxy2 + y2sx2 + x2];
					//t = itrilerp2(dx1, dx2, dy1, dy2, dz1, dz2, q1, q2, q3, q4, q5, q6, q7, q8);
					//t = itrilerp(tx, ty, tz, x1, x2, y1, y2, z1, z2, q1, q2, q3, q4, q5, q6, q7, q8);
					t = dz2*(dy2*dx2*q1 + dy2*dx1*q2 + dy1*dx2*q3 + dy1*dx1*q4) + dz1*(dy2*dx2*q5 + dy2*dx1*q6 + dy1*dx2*q7 + dy1*dx1*q8);
					//t = 1;

				}
				else
					t = 0;
				s = h_s[k*sxy + j*sx + i];

				sqrSum += (double)t*t;
				corrSum += (double)s*t;
			}
		}
	}
	return (corrSum / sqrt(sqrSum));
}

void affinetransformcpu_old(float *h_s,
	float *h_t,// source stack
	float *aff,
	int sx,
	int sy,
	int sz,
	int sx2,
	int sy2,
	int sz2
	){
	float ix, iy, iz, tx, ty, tz;
	int x1, y1, z1, x2, y2, z2;
	float dx1, dy1, dz1, dx2, dy2, dz2;
	float q1, q2, q3, q4, q5, q6, q7, q8;
	float t;
	int sxy = sx*sy, sxy2 = sx2*sy2, z1sxy2, z2sxy2, y1sx2, y2sx2;
	int syz = sy*sz, syz2 = sy2*sz2;
	for (int i = 0; i < sx; i++){
		ix = (float)i;
		for (int j = 0; j < sy; j++){
			iy = (float)j;
			for (int k = 0; k < sz; k++){
				iz = (float)k;
				tx = aff[0] * ix + aff[1] * iy + aff[2] * iz + aff[3];
				ty = aff[4] * ix + aff[5] * iy + aff[6] * iz + aff[7];
				tz = aff[8] * ix + aff[9] * iy + aff[10] * iz + aff[11];
				x1 = (int)tx; y1 = (int)ty; z1 = (int)tz;
				x2 = x1 + 1; y2 = y1 + 1; z2 = z1 + 1;

				dx1 = tx - (float)x1; dy1 = ty - (float)y1; dz1 = tz - (float)z1;
				dx2 = 1 - dx1; dy2 = 1 - dy1; dz2 = 1 - dz1;
				if (x1 >= 0 && y1 >= 0 && z1 >= 0 && x2 < sx2 && y2 < sy2 && z2 < sz2){
					// [i*sy*sz + j*sz + k]
					z1sxy2 = z1*sxy2;
					z2sxy2 = z2*sxy2;
					y1sx2 = y1*sx2;
					y2sx2 = y2*sx2;

					q1 = h_t[z1sxy2 + y1sx2 + x1];
					q2 = h_t[z1sxy2 + y1sx2 + x2];
					q3 = h_t[z1sxy2 + y2sx2 + x1];
					q4 = h_t[z1sxy2 + y2sx2 + x2];
					q5 = h_t[z2sxy2 + y1sx2 + x1];
					q6 = h_t[z2sxy2 + y1sx2 + x2];
					q7 = h_t[z2sxy2 + y2sx2 + x1];
					q8 = h_t[z2sxy2 + y2sx2 + x2];
					t = itrilerp2(dx1, dx2, dy1, dy2, dz1, dz2, q1, q2, q3, q4, q5, q6, q7, q8);
					//t = itrilerp(tx, ty, tz, x1, x2, y1, y2, z1, z2, q1, q2, q3, q4, q5, q6, q7, q8);
					//t = dz2*(dy2*dx2*q1 + dy2*dx1*q2 + dy1*dx2*q3 + dy1*dx1*q4) + dz1*(dy2*dx2*q5 + dy2*dx1*q6 + dy1*dx2*q7 + dy1*dx1*q8);
				}
				else
					t = 0;
				h_s[k*sxy + j*sx + j] = t;
			}
		}
	}
}


double corrfunccpu2(float *h_s,
	float *h_t,// source stack
	float *aff,
	int sx,
	int sy,
	int sz,
	int sx2,
	int sy2,
	int sz2
	){
	const float r0 = aff[0], r1 = aff[1], r2 = aff[2], r3 = aff[3], r4 = aff[4], r5 = aff[5],
		r6 = aff[6], r7 = aff[7], r8 = aff[8], r9 = aff[9], r10 = aff[10], r11 = aff[11];

	double sqrSum = 0, corrSum = 0;
	float ix, iy, iz, tx, ty, tz;
	int x1, y1, z1, x2, y2, z2;
	float dx1, dy1, dz1, dx2, dy2, dz2;
	float q1, q2, q3, q4, q5, q6, q7, q8;
	float s, t;
	int syz = sy*sz, syz2 = sy2*sz2, x1syz2, x2syz2, y1sz2, y2sz2;
	for (int i = 0; i < sx; i++){
		ix = (float)i;
		for (int j = 0; j < sy; j++){
			iy = (float)j;
			for (int k = 0; k < sz; k++){
				iz = (float)k;

				tx = r0 * ix + r1 * iy + r2 * iz + r3;
				ty = r4 * ix + r5 * iy + r6 * iz + r7;
				tz = r8 * ix + r9 * iy + r10 * iz + r11;

				x1 = (int)tx; y1 = (int)ty; z1 = (int)tz;
				x2 = x1 + 1; y2 = y1 + 1; z2 = z1 + 1;

				dx1 = tx - (float)x1; dy1 = ty - (float)y1; dz1 = tz - (float)z1;
				dx2 = 1 - dx1; dy2 = 1 - dy1; dz2 = 1 - dz1;
				if (x1 >= 0 && y1 >= 0 && z1 >= 0 && x2 < sx2 && y2 < sy2 && z2 < sz2){
					// [i*sy*sz + j*sz + k]
					x1syz2 = x1*syz2;
					x2syz2 = x2*syz2;
					y1sz2 = y1*sz2;
					y2sz2 = y2*sz2;

					q1 = h_t[x1syz2 + y1sz2 + z1];
					q2 = h_t[x2syz2 + y1sz2 + z1];
					q3 = h_t[x1syz2 + y2sz2 + z1];
					q4 = h_t[x2syz2 + y2sz2 + z1];
					q5 = h_t[x1syz2 + y1sz2 + z2];
					q6 = h_t[x2syz2 + y1sz2 + z2];
					q7 = h_t[x1syz2 + y2sz2 + z2];
					q8 = h_t[x2syz2 + y2sz2 + z2];
					t = itrilerp2(dx1, dx2, dy1, dy2, dz1, dz2, q1, q2, q3, q4, q5, q6, q7, q8);

				}
				else
					t = 0;
				s = h_s[i*syz + j*sz + k];

				sqrSum += (double)t*t;
				corrSum += (double)s*t;
			}
		}
	}
	return (corrSum / sqrt(sqrSum));
}


void affinetransformcpu(float *h_s,
	float *h_t,// source stack
	float *aff,
	int sx,
	int sy,
	int sz,
	int sx2,
	int sy2,
	int sz2
	){
	float ix, iy, iz, tx, ty, tz;
	int x1, y1, z1, x2, y2, z2;
	float dx1, dy1, dz1, dx2, dy2, dz2;
	float q1, q2, q3, q4, q5, q6, q7, q8;
	float t;
	int syz = sy*sz, syz2 = sy2*sz2, x1syz2, x2syz2, y1sz2, y2sz2;
	for (int i = 0; i < sx; i++){
		ix = (float)i;
		for (int j = 0; j < sy; j++){
			iy = (float)j;
			for (int k = 0; k < sz; k++){
				iz = (float)k;
				tx = aff[0] * ix + aff[1] * iy + aff[2] * iz + aff[3];
				ty = aff[4] * ix + aff[5] * iy + aff[6] * iz + aff[7];
				tz = aff[8] * ix + aff[9] * iy + aff[10] * iz + aff[11];
				x1 = (int)tx; y1 = (int)ty; z1 = (int)tz;
				x2 = x1 + 1; y2 = y1 + 1; z2 = z1 + 1;

				dx1 = tx - (float)x1; dy1 = ty - (float)y1; dz1 = tz - (float)z1;
				dx2 = 1 - dx1; dy2 = 1 - dy1; dz2 = 1 - dz1;
				if (x1 >= 0 && y1 >= 0 && z1 >= 0 && x2 < sx2 && y2 < sy2 && z2 < sz2){
					// [i*sy*sz + j*sz + k]
					x1syz2 = x1*syz2;
					x2syz2 = x2*syz2;
					y1sz2 = y1*sz2;
					y2sz2 = y2*sz2;

					q1 = h_t[x1syz2 + y1sz2 + z1];
					q2 = h_t[x2syz2 + y1sz2 + z1];
					q3 = h_t[x1syz2 + y2sz2 + z1];
					q4 = h_t[x2syz2 + y2sz2 + z1];
					q5 = h_t[x1syz2 + y1sz2 + z2];
					q6 = h_t[x2syz2 + y1sz2 + z2];
					q7 = h_t[x1syz2 + y2sz2 + z2];
					q8 = h_t[x2syz2 + y2sz2 + z2];
					t = itrilerp2(dx1, dx2, dy1, dy2, dz1, dz2, q1, q2, q3, q4, q5, q6, q7, q8);

				}
				else
					t = 0;
				h_s[i*syz + j*sz + k] = t;
			}
		}
	}
}

// CPU
template <class T>
void flipcpu(T *h_odata, T *h_idata, long long int sx, long long int sy, long long int sz) {
	for (long long int i = 0; i < sx; i++) {
		for (long long int j = 0; j < sy; j++) {
			for (long long int k = 0; k < sz; k++) {
				//d_odata[k*sy*sx + j*sx + i] = d_idata[(sz - k - 1) *sy*sx + (sy - j - 1)*sx + (sx - i - 1)];
				h_odata[i*sy*sz + j*sz + k] = h_idata[(sx - i - 1) *sy*sz + (sy - j - 1)*sz + (sz - k - 1)];
			}
		}
	}
}
template void flipcpu<int>(int *h_odata, int *h_idata, long long int sx, long long int sy, long long int sz);
template void flipcpu<float>(float *h_odata, float *h_idata, long long int sx, long long int sy, long long int sz);
template void flipcpu<double>(double *h_odata, double *h_idata, long long int sx, long long int sy, long long int sz);

template <class T>
void padPSFcpu(T *h_odata, T *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2, 
	long long int sy2, long long int sz2){
	long long int sox, soy, soz;
	sox = sx2 / 2; soy = sy2 / 2; soz = sz2 / 2;
	long long int dx, dy, dz;
	for (long long int x = 0; x < sx; x++) {
		for (long long int y = 0; y < sy; y++) {
			for (long long int z = 0; z < sz; z++) {
				dx = x - sox; dy = y - soy; dz = z - soz;
				if (dx < 0) dx += sx;
				if (dy < 0) dy += sy;
				if (dz < 0) dz += sz;
				//d_PaddedPSF[dz][dy][dx] = d_PSF[z][y][x]
				if (dx >= 0 && dx < sx && dy >= 0 && dy < sy && dz >= 0 && dz < sz) {
					//d_odata[dz*sy*sx + dy*sx + dx] = d_idata[z*sy2*sx2 + y*sx2 + x];
					h_odata[dx*sy*sz + dy*sz + dz] = h_idata[x*sy2*sz2 + y*sz2 + z];
				}
			}
		}
	}
}
template void
padPSFcpu<int>(int *h_odata, int *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);
template void
padPSFcpu<float>(float *h_odata, float *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);
template void
padPSFcpu<double>(double *h_odata, double *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);

template <class T>
void padstackcpu(T *h_odata, T *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2){
	long long int sox, soy, soz;
	sox = (sx - sx2) / 2;
	soy = (sy - sy2) / 2;
	soz = (sz - sz2) / 2;
	long long int x, y, z;
	for (long long int dx = 0; dx < sx; dx++) {
		for (long long int dy = 0; dy < sy; dy++) {
			for (long long int dz = 0; dz < sz; dz++) {
				if (dx < sox) {
					x = 0;
				}
				if (dy < soy) {
					y = 0;
				}
				if (dz < soz) {
					z = 0;
				}
				if (dx >= sox && dx < (sox + sx2)) {
					x = dx - sox;
				}
				if (dy >= soy && dy < (soy + sy2)) {
					y = dy - soy;
				}
				if (dz >= soz && dz < (soz + sz2)) {
					z = dz - soz;
				}
				if (dx >= (sox + sx2)) {
					x = sx2 - 1;
				}
				if (dy >= (soy + sy2)) {
					y = sy2 - 1;
				}
				if (dz >= (soz + sz2)) {
					z = sz2 - 1;
				}
				//d_odata[dz*sy*sx + dy*sx + dx] = d_idata[z*sy2*sx2 + y*sx2 + x];
				h_odata[dx*sy*sz + dy*sz + dz] = h_idata[x*sy2*sz2 + y*sz2 + z];
			}
		}
	}
}
template void
padstackcpu<int>(int *h_odata, int *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);
template void
padstackcpu<float>(float *h_odata, float *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);
template void
padstackcpu<double>(double *h_odata, double *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);

template <class T>
void cropcpu(T *h_odata, T *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2){
	long long int sox, soy, soz;
	sox = (sx2 - sx) / 2;
	soy = (sy2 - sy) / 2;
	soz = (sz2 - sz) / 2;
	long long int dx, dy, dz;
	for (long long int x = 0; x < sx; x++) {
		for (long long int y = 0; y < sy; y++) {
			for (long long int z = 0; z < sz; z++) {
				dx = sox + x; dy = soy + y; dz = soz + z;
				//d_odata[z*sy*sx + y*sx + x] = d_idata[dz*sy2*sx2 + dy*sx2 + dx];
				h_odata[x*sy*sz + y*sz + z] = h_idata[dx*sy2*sz2 + dy*sz2 + dz];
			}
		}
	}
}
template void
cropcpu<int>(int *h_odata, int *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);
template void
cropcpu<float>(float *h_odata, float *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);
template void
cropcpu<double>(double *h_odata, double *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);

template <class T>
void cropcpu2(T *h_odata, T *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2, long long int sox, long long int soy, long long int soz) {
	long long int dx, dy, dz;
	for (long long int x = 0; x < sx; x++) {
		for (long long int y = 0; y < sy; y++) {
			for (long long int z = 0; z < sz; z++) {
				dx = sox + x; dy = soy + y; dz = soz + z;
				h_odata[z*sy*sx + y*sx + x] = h_idata[dz*sy2*sx2 + dy*sx2 + dx];
				//h_odata[x*sy*sz + y*sz + z] = h_idata[dx*sy2*sz2 + dy*sz2 + dz];
			}
		}
	}
}
template void
cropcpu2<int>(int *h_odata, int *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2, long long int sox, long long int soy, long long int soz);
template void
cropcpu2<float>(float *h_odata, float *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2, long long int sox, long long int soy, long long int soz);
template void
cropcpu2<double>(double *h_odata, double *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2, long long int sox, long long int soy, long long int soz);

template <class T>
void alignsize3Dcpu(T *h_odata, T *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2) {
	long long int sox, soy, soz;
	sox = (sx - sx2) / 2;
	soy = (sy - sy2) / 2;
	soz = (sz - sz2) / 2;
	long long int x, y, z;
	for (long long int dx = 0; dx < sx; dx++) {
		for (long long int dy = 0; dy < sy; dy++) {
			for (long long int dz = 0; dz < sz; dz++) {
				x = dx - sox;
				y = dy - soy;
				z = dz - soz;
				if ((x < 0) || (y < 0) || (z < 0) || (x >= sx2) || (y >= sy2) || (z >= sz2))
					h_odata[dx*sy*sz + dy*sz + dz] = 0;
				else
					h_odata[dx*sy*sz + dy*sz + dz] = h_idata[x*sy2*sz2 + y*sz2 + z];
			}
		}
	}
}
template void alignsize3Dcpu<int>(int *h_odata, int *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);
template void alignsize3Dcpu<float>(float *h_odata, float *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);
template void alignsize3Dcpu<double>(double *h_odata, double *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);

extern "C"
void genOTFcpu(fftwf_complex *h_odata, float *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2, bool normFlag) {
	long long int totalSizeIn = sx2 * sy2 * sz2;
	long long int totalSizeOut = sx * sy * sz;
	long long int totalSizeMax = (totalSizeIn > totalSizeOut) ? totalSizeIn : totalSizeOut;
	float *h_temp = (float *)malloc(totalSizeMax * sizeof(float));
	if (normFlag) {
		double sumValue = sumcpu(h_idata, sx2 * sy2 * sz2);
		multivaluecpu(h_temp, h_idata, (float)(1 / sumValue), sx2 * sy2 * sz2);
	}
	else
		memcpy(h_temp, h_idata, totalSizeIn * sizeof(float));
	
	if((sx<sx2)||(sy<sy2)||(sz<sz2)){
		alignsize3Dcpu((float *)h_odata, h_temp, sx, sy, sz, sx2, sy2, sz2);
		padPSFcpu(h_temp, (float *)h_odata, sx, sy, sz, sx, sy, sz);
	}
	else {
		padPSFcpu((float *)h_odata, h_temp, sx, sy, sz, sx2, sy2, sz2);
		memcpy(h_temp, h_odata, totalSizeOut * sizeof(float));
	}
	fftwf_plan image2Spectrum = fftwf_plan_dft_r2c_3d(sx, sy, sz, h_temp, h_odata,  FFTW_MEASURE);
	fftwf_execute(image2Spectrum);
	free(h_temp);
	fftwf_destroy_plan(image2Spectrum);
}

// GPU
template <class T>
void flipgpu(T *d_odata, T *d_idata, long long int sx, long long int sy, long long int sz) {
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sx, blockSize3Dx), iDivUp(sy, blockSize3Dy), iDivUp(sz, blockSize3Dz));
	flipgpukernel<T> << <grids, threads >> >(d_odata, d_idata, sx, sy, sz);
	hipDeviceSynchronize();
}
template void flipgpu<int>(int *d_odata, int *d_idata, long long int sx, long long int sy, long long int sz);
template void flipgpu<float>(float *d_odata, float *d_idata, long long int sx, long long int sy, long long int sz);
template void flipgpu<double>(double *d_odata, double *d_idata, long long int sx, long long int sy, long long int sz);

template <class T>
void padPSFgpu(T *d_odata, T *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2) {
	assert(d_odata != d_idata);
	long long int sox, soy, soz;
	sox = sx2 / 2; soy = sy2 / 2; soz = sz2 / 2;
	hipMemset(d_odata, 0, sx*sy*sz * sizeof(T));
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sx2, threads.x), iDivUp(sy2, threads.y), iDivUp(sz2, threads.z));
	padPSFgpukernel<T> << <grids, threads >> >(d_odata, d_idata, sx, sy, sz, sx2, sy2, sz2, sox, soy, soz);
	hipDeviceSynchronize();
}
template void
padPSFgpu<int>(int *d_odata, int *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);
template void
padPSFgpu<float>(float *d_odata, float *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);
template void
padPSFgpu<double>(double *d_odata, double *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);

template <class T>
void padstackgpu(T *d_odata, T *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2) {
	assert(d_odata != d_idata);
	long long int sox, soy, soz;
	sox = (sx - sx2) / 2;
	soy = (sy - sy2) / 2;
	soz = (sz - sz2) / 2;
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sx, threads.x), iDivUp(sy, threads.y), iDivUp(sz, threads.z));
	padstackgpukernel<T> << < grids, threads >> > (d_odata, d_idata, sx, sy, sz, sx2, sy2, sz2, sox, soy, soz);
	hipDeviceSynchronize();
}
template void
padstackgpu<int>(int *d_odata, int *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);
template void
padstackgpu<float>(float *d_odata, float *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);
template void
padstackgpu<double>(double *d_odata, double *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);

template <class T>
void cropgpu(T *d_odata, T *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2) {
	assert(d_odata != d_idata);
	long long int sox, soy, soz;
	sox = (sx2 - sx) / 2;
	soy = (sy2 - sy) / 2;
	soz = (sz2 - sz) / 2;
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sx, threads.x), iDivUp(sy, threads.y), iDivUp(sz, threads.z));
	cropgpukernel<T> <<< grids, threads >>> (d_odata, d_idata, sx, sy, sz, sx2, sy2, sz2, sox, soy, soz);
	hipDeviceSynchronize();
}
template void
cropgpu<int>(int *d_odata, int *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);
template void
cropgpu<float>(float *d_odata, float *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);
template void
cropgpu<double>(double *d_odata, double *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);

template <class T>
void cropgpu2(T *d_odata, T *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2, long long int sox, long long int soy, long long int soz) {
	assert(d_odata != d_idata);
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sz, threads.x), iDivUp(sy, threads.y), iDivUp(sx, threads.z));
	cropgpukernel<T> <<< grids, threads >>> (d_odata, d_idata, sz, sy, sx, sz2, sy2, sx2, soz, soy, sox);
	hipDeviceSynchronize();
}
template void
cropgpu2<int>(int *d_odata, int *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2, long long int sox, long long int soy, long long int soz);
template void
cropgpu2<float>(float *d_odata, float *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2, long long int sox, long long int soy, long long int soz);
template void
cropgpu2<double>(double *d_odata, double *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2, long long int sox, long long int soy, long long int soz);


template <class T>
void alignsize3Dgpu(T *d_odata, T *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2) {
	assert(d_odata != d_idata);
	long long int sox, soy, soz;
	sox = (sx - sx2) / 2;
	soy = (sy - sy2) / 2;
	soz = (sz - sz2) / 2;
	dim3 threads(blockSize3Dx, blockSize3Dy, blockSize3Dz);
	dim3 grids(iDivUp(sx, threads.x), iDivUp(sy, threads.y), iDivUp(sz, threads.z));
	alignsize3Dgpukernel<T> << < grids, threads >> > (d_odata, d_idata, sx, sy, sz, sx2, sy2, sz2, sox, soy, soz);
	hipDeviceSynchronize();
}
template void alignsize3Dgpu<int>(int *d_odata, int *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);
template void alignsize3Dgpu<float>(float *d_odata, float *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);
template void alignsize3Dgpu<double>(double *d_odata, double *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2);

// Registration variables: 2D
static float *d_img2D = NULL;
static float *h_aff2D;
static long long int imx2D1, imy2D1, imx2D2, imy2D2;
static float valueStatic2D;
static int itNumStatic2D;

// Registration variables: 3D
static float *d_imgStatic = NULL;
static float valueStatic;
static long long int sxStatic1, syStatic1, szStatic1, sxStatic2, syStatic2, szStatic2;
static float *affCoef;
static int itNumStatic, dofNum;
static bool dof9Flag;
static float *h_s3D = NULL, *h_t3D = NULL;

float costfunc2D(float *x) {
	h_aff2D[0] = x[1], h_aff2D[1] = x[2], h_aff2D[2] = x[3];
	h_aff2D[3] = x[4], h_aff2D[4] = x[5], h_aff2D[5] = x[6];
	float costValue = corrfunc2D(d_img2D, valueStatic2D, h_aff2D, imx2D1, imy2D1, imx2D2, imy2D2);
	itNumStatic2D += 1;
	return -costValue;
}
extern "C"
int affinetrans2d0(float *h_odata, float *iTmx, float *h_idata, long long int sx, long long int sy, long long int sx2, long long int sy2) {
	return 0;
}
extern "C"
// bug in affinetrans2d1 
int affinetrans2d1(float *h_odata, float *iTmx, float *h_idata, long long int sx, long long int sy, long long int sx2, long long int sy2) {
	// total pixel count for each images
	long long int totalSize1 = sx*sy;
	long long int totalSize2 = sx2*sx2;

	float *d_imgTemp = NULL;
	hipMalloc((void **)&d_imgTemp, totalSize1 * sizeof(float));
	cudaCheckErrors("****Memory allocating fails... GPU out of memory !!!!*****\n");

	hipChannelFormatDesc channelDesc2D =
		hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray *d_Array2D;
	hipMallocArray(&d_Array2D, &channelDesc2D, sx2, sy2);
	cudaCheckErrors("****Memory array allocating fails... GPU out of memory !!!!*****\n");
	CopyTranMatrix(iTmx, 6 * sizeof(float));
	hipMemcpyToArray(d_Array2D, 0, 0, h_idata, totalSize2 * sizeof(float), hipMemcpyHostToDevice);
	BindTexture2D(d_Array2D, channelDesc2D);
	affineTransform2D(d_imgTemp, sx, sy, sx2, sy2);
	UnbindTexture2D;
	hipMemcpy(h_odata, d_imgTemp, totalSize1 * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_imgTemp);
	hipFreeArray(d_Array2D);
	return 0;
}

extern "C"
int reg2d_phasor0(long long int *shiftXY, float *h_img1, float *h_img2, long long int sx, long long int sy) {
	return 0;
}
extern "C"
int reg2d_phasor1(long long int *shiftXY, float *d_img1, float *d_img2, long long int sx, long long int sy) {
	int totalSize = sx * sy;
	int totalSizeSpectrum = sy*(sx / 2 + 1); // in complex floating format
	fComplex *d_Spectrum1 = NULL, *d_Spectrum2 = NULL;
	hipMalloc((void **)&d_Spectrum1, totalSizeSpectrum * sizeof(fComplex));
	hipMalloc((void **)&d_Spectrum2, totalSizeSpectrum * sizeof(fComplex));
	hipfftHandle
		fftPlanFwd,
		fftPlanInv;
	hipfftPlan2d(&fftPlanFwd, sy, sx, HIPFFT_R2C);
	hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_img1, (hipfftComplex *)d_Spectrum2);
	conj3Dgpu(d_Spectrum1, d_Spectrum2, sy, (sx / 2 + 1), 1);
	hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_img2, (hipfftComplex *)d_Spectrum2);
	// multiplication and normalization
	multicomplexnorm3Dgpu(d_Spectrum2, d_Spectrum1, d_Spectrum2, sy, (sx / 2 + 1), 1);
	hipfftDestroy(fftPlanFwd);
	hipfftPlan2d(&fftPlanInv, sy, sx, HIPFFT_C2R);
	float *d_phasor1 = (float *)d_Spectrum1;
	hipfftExecC2R(fftPlanInv, (hipfftComplex *)d_Spectrum2, (hipfftReal *)d_phasor1);
	hipfftDestroy(fftPlanInv);
	size_t corXYZ[3];
	float *d_phasor2 = (float *)d_Spectrum2;
	circshiftgpu(d_phasor2, d_phasor1, sx, sy, 1, round(sx / 2), round(sy / 2), 0);
	float peakValue = max3Dgpu(&corXYZ[0], d_phasor2, sx, sy, 1);
	shiftXY[0] = long long int(corXYZ[0]) - sx / 2;
	shiftXY[1] = long long int(corXYZ[1]) - sy / 2;
	hipFree(d_Spectrum1);
	hipFree(d_Spectrum2);

	// compare 4 cases based on cross-correlation
	long long int shiftX = shiftXY[0];
	long long int shiftY = shiftXY[1];
	long long int xabs = abs(shiftX), yabs = abs(shiftY);
	long long int beta = 4; // threshold value: only if shift is more than 1/beta of the image size
	if ((xabs >(sx / beta)) || (yabs >(sy / beta))) {
		float *d_imgT = NULL, *d_crop1 = NULL, *d_crop2 = NULL;
		long long int sizex1, sizex2, sizey1, sizey2, sizez1, sizez2, sizex, sizey, sizez, sizeMaxCrop;
		sizeMaxCrop = totalSize;
		hipMalloc((void **)&d_imgT, totalSize * sizeof(float));
		hipMalloc((void **)&d_crop1, sizeMaxCrop * sizeof(float));
		hipMalloc((void **)&d_crop2, sizeMaxCrop * sizeof(float));
		circshiftgpu(d_imgT, d_img2, sx, sy, 1, -shiftX, -shiftY, 0);
		// encode the 8 cases as for loop
		long long int imSizeCropx[2], imSizeCropy[2], imSizeCropz[2];
		long long int imox[2], imoy[2], imoz[2];
		// index 0 records original shifts, index 1 switches the shift to the opposite case.  
		imSizeCropx[0] = sx - xabs; imSizeCropx[1] = xabs;
		if (shiftX > 0) {
			imox[0] = 0; imox[1] = sx - xabs;
		}
		else {
			imox[0] = xabs; imox[1] = 0;
		}
		imSizeCropy[0] = sy - yabs; imSizeCropy[1] = yabs;
		if (shiftY > 0) {
			imoy[0] = 0; imoy[1] = sy - yabs;
		}
		else {
			imoy[0] = yabs; imoy[1] = 0;
		}
		int indx = 0, indy = 0;
		float ccMax = -3, ccNow = 0;
		for (int i = 0; i < 2; i++) {
			if (imSizeCropx[i] >(sx / beta)) {
				for (int j = 0; j < 2; j++) {
					if (imSizeCropy[j] >(sy / beta)) {
						cropgpu2(d_crop1, d_img1, imSizeCropx[i], imSizeCropy[j], 1, sx, sy, 1, imox[i], imoy[j], 0);
						cropgpu2(d_crop2, d_imgT, imSizeCropx[i], imSizeCropy[j], 1, sx, sy, 1, imox[i], imoy[j], 0);
						ccNow = zncc1(d_crop1, d_crop2, imSizeCropx[i], imSizeCropy[j], 1);
						if (ccMax < ccNow) {
							ccMax = ccNow;
							indx = i;
							indy = j;
						}
					}
				}
			}
		}
		// if ind ==1, flip the coordinates
		if (indx == 1) {
			if (shiftX > 0)
				shiftXY[0] = shiftX - sx;
			else
				shiftXY[0] = shiftX + sx;
		}
		if (indy == 1) {
			if (shiftY > 0)
				shiftXY[1] = shiftY - sy;
			else
				shiftXY[1] = shiftY + sy;
		}
		hipFree(d_imgT);
		hipFree(d_crop1);
		hipFree(d_crop2);
	}
	return 0;
}

extern "C"
int reg2d_affine0(float *h_reg, float *iTmx, float *h_img1, float *h_img2, long long int sx, long long int sy,
	long long int sx2, long long int sy2, int affMethod, bool flagTmx, float FTOL, int itLimit, float *regRecords) {
	// **** CPU affine registration for 2D images ***
	return 0;
}
extern "C"
int reg2d_affine1(float *h_reg, float *iTmx, float *h_img1, float *h_img2, long long int sx, long long int sy, 
	long long int sx2, long long int sy2, int affMethod, bool flagTmx, float FTOL, int itLimit, float *records) {
	// **** GPU affine registration for 2D images ***
	/*
	*** flagTmx:
	true : use iTmx as input matrix;
	false: default;

	*** records: 8 element array
	[1] -[3]: initial ZNCC (zero-normalized cross-correlation, negtive of the cost function), intermediate ZNCC, optimized ZNCC;
	[4] -[7]: single sub iteration time (in ms), total number of sub iterations, iteralation time (in s), whole registration time (in s);
	*/
	imx2D1 = sx; imy2D1 = sy;
	imx2D2 = sx2; imy2D2 = sy2;

	// total pixel count for each images
	long long int totalSize1 = imx2D1*imy2D1;
	long long int totalSize2 = imx2D2*imy2D2;
	long long int totalSizeMax = (totalSize1 > totalSize2) ? totalSize1 : totalSize2;

	// ****************** Processing Starts***************** //
	// variables for memory and time cost records
	clock_t start, end, ctime1, ctime2, ctime3;
	start = clock();
	int iter;
	float fret;
	int DIM2D = 6;
	h_aff2D = (float *)malloc(DIM2D * sizeof(float));
	static float *p2D = (float *)malloc((DIM2D + 1) * sizeof(float));
	float **xi2D;
	xi2D = matrix(1, DIM2D, 1, DIM2D);

	float *h_imgT = (float *)malloc(totalSizeMax * sizeof(float));
	hipMalloc((void **)&d_img2D, totalSize1 * sizeof(float));
	cudaCheckErrors("****Memory allocating fails... GPU out of memory !!!!*****\n");

	hipChannelFormatDesc channelDesc2D =
		hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray *d_Array2D;
	hipMallocArray(&d_Array2D, &channelDesc2D, imx2D2, imy2D2);
	cudaCheckErrors("****Memory array allocating fails... GPU out of memory !!!!*****\n");

	if (flagTmx) {
		memcpy(h_aff2D, iTmx, DIM2D * sizeof(float));
	}
	else {
		h_aff2D[0] = 1, h_aff2D[1] = 0, h_aff2D[2] = (imx2D2 - imx2D1) / 2;
		h_aff2D[3] = 0, h_aff2D[4] = 1, h_aff2D[5] = (imy2D2 - imy2D1) / 2;
	}
	p2D[0] = 0;
	p2D[1] = h_aff2D[0], p2D[2] = h_aff2D[1], p2D[3] = h_aff2D[2];
	p2D[4] = h_aff2D[3], p2D[5] = h_aff2D[4], p2D[6] = h_aff2D[5];
	for (int i = 1; i <= DIM2D; i++)
		for (int j = 1; j <= DIM2D; j++)
			xi2D[i][j] = (i == j ? 1.0 : 0.0);

	float meanValue = (float)sumcpu(h_img1, totalSize1) / totalSize1;
	addvaluecpu(h_imgT, h_img1, -meanValue, totalSize1);
	multicpu(h_reg, h_imgT, h_imgT, totalSize1);
	double sumSqrA = sumcpu(h_reg, totalSize1);
	valueStatic2D = float(sqrt(sumSqrA));
	if (valueStatic2D == 0) {
		fprintf(stderr, "*** SD of image 1 is zero, empty image input **** \n");
		exit(1);
	}
	hipMemcpy(d_img2D, h_imgT, totalSize1 * sizeof(float), hipMemcpyHostToDevice);

	meanValue = (float)sumcpu(h_img2, totalSize2) / totalSize2;
	addvaluecpu(h_imgT, h_img2, -meanValue, totalSize2);
	hipMemcpyToArray(d_Array2D, 0, 0, h_imgT, totalSize2 * sizeof(float), hipMemcpyHostToDevice);
	BindTexture2D(d_Array2D, channelDesc2D);
	cudaCheckErrors("****Fail to bind 2D texture!!!!*****\n");
	itNumStatic2D = 0;
	ctime1 = clock();
	records[1] = -costfunc2D(p2D);
	ctime2 = clock();
	if (affMethod > 0) {
		powell(p2D, xi2D, DIM2D, FTOL, &iter, &fret, costfunc2D, &itNumStatic2D, itLimit);
		memcpy(iTmx, h_aff2D, DIM2D * sizeof(float));
	}
	UnbindTexture2D;
	ctime3 = clock();

	hipMemcpyToArray(d_Array2D, 0, 0, h_img2, totalSize2 * sizeof(float), hipMemcpyHostToDevice);
	BindTexture2D(d_Array2D, channelDesc2D);
	affineTransform2D(d_img2D, imx2D1, imy2D1, imx2D2, imy2D2);
	UnbindTexture2D;
	hipMemcpy(h_reg, d_img2D, totalSize1 * sizeof(float), hipMemcpyDeviceToHost);

	records[3] = -fret;
	records[4] = (float)(ctime2 - ctime1);
	records[5] = itNumStatic2D;
	records[6] = (float)(ctime3 - ctime2) / CLOCKS_PER_SEC;
	free(p2D);
	free(h_aff2D);
	free_matrix(xi2D, 1, DIM2D, 1, DIM2D);
	free(h_imgT);
	hipFree(d_img2D);
	hipFreeArray(d_Array2D);

	end = clock();
	records[7] = (float)(end - start) / CLOCKS_PER_SEC;
	return 0;
}

extern "C"
int affinetrans3d0(float *h_odata, float *iTmx, float *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2) {
	// cpu
	return 0;
}
extern "C"
int affinetrans3d1(float *d_odata, float *iTmx, float *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2) {
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipArray *d_ArrayTemp;
	hipMalloc3DArray(&d_ArrayTemp, &channelDesc, make_hipExtent(sx2, sy2, sz2));
	hipDeviceSynchronize();
	cudaCheckErrors("****GPU array memory allocating fails... GPU out of memory !!!!*****\n");
	cudacopydevicetoarray(d_ArrayTemp, channelDesc, d_idata, sx2, sy2, sz2);
	BindTexture(d_ArrayTemp, channelDesc);
	CopyTranMatrix(iTmx, NDIM * sizeof(float));
	affineTransform(d_odata, sx, sy, sz, sx2, sy2, sz2);
	UnbindTexture();
	hipFreeArray(d_ArrayTemp);
	return 0;
}
extern "C"
int affinetrans3d2(float *d_odata, float *iTmx, float *h_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2) {
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipArray *d_ArrayTemp;
	hipMalloc3DArray(&d_ArrayTemp, &channelDesc, make_hipExtent(sx2, sy2, sz2));
	hipDeviceSynchronize();
	cudaCheckErrors("****GPU array memory allocating fails... GPU out of memory !!!!*****\n");
	cudacopyhosttoarray(d_ArrayTemp, channelDesc, h_idata, sx2, sy2, sz2);
	BindTexture(d_ArrayTemp, channelDesc);
	CopyTranMatrix(iTmx, NDIM * sizeof(float));
	affineTransform(d_odata, sx, sy, sz, sx2, sy2, sz2);
	UnbindTexture();
	hipFreeArray(d_ArrayTemp);
	return 0;
}

float costfunc(float *x) {
	if (dof9Flag) {
		dof9tomatrix(affCoef, x, dofNum);
	}
	else {
		p2matrix(affCoef, x);
	}
	float costValue = corrfunc(d_imgStatic, valueStatic, affCoef, sxStatic1, syStatic1, szStatic1, sxStatic2, syStatic2, szStatic2);

	itNumStatic += 1;
	return -costValue;
}

float costfunccpu(float *x) { // **** this function does not work correctly
	if (dof9Flag) {
		dof9tomatrix(affCoef, x, dofNum);
	}
	else {
		p2matrix(affCoef, x);
	}

	double costValue = corrfunccpu2(h_s3D, h_t3D, affCoef, sxStatic1, syStatic1, szStatic1, sxStatic2, syStatic2, szStatic2);

	itNumStatic += 1;
	return (float)(-costValue / valueStatic);
}

extern "C"
float zncc0(float *h_img1, float *h_img2, long long int sx, long long int sy, long long int sz) {
	return 0;
}
extern "C"
float zncc1(float *d_img1, float *d_img2, long long int sx, long long int sy, long long int sz) {
	// d_img1, d_img2 value change after calculation
	float znccValue = -2.0;
	long long int totalSize = sx*sy*sz;
	float *d_imgT = NULL;
	hipMalloc((void **)&d_imgT, totalSize * sizeof(float));
	cudaCheckErrors("****GPU memory allocating fails... GPU out of memory !!!!*****\n");
	double sumImg1 = 0, sumImg2 = 0, sumST = 0, sumSS = 0, sumTT = 0;
	sumImg1 = sum3Dgpu(d_img1, sx, sy, sz);
	sumImg2 = sum3Dgpu(d_img2, sx, sy, sz);
	addvaluegpu(d_img1, d_img1, -float(sumImg1) / float(totalSize), sx, sy, sz);
	addvaluegpu(d_img2, d_img2, -float(sumImg2) / float(totalSize), sx, sy, sz);
	multi3Dgpu(d_imgT, d_img1, d_img2, sx, sy, sz);
	sumST = sum3Dgpu(d_imgT, sx, sy, sz);
	multi3Dgpu(d_imgT, d_img1, d_img1, sx, sy, sz);
	sumTT = sum3Dgpu(d_imgT, sx, sy, sz);
	multi3Dgpu(d_imgT, d_img2, d_img2, sx, sy, sz);
	sumSS = sum3Dgpu(d_imgT, sx, sy, sz);
	hipFree(d_imgT);
	float b = float(sqrt(sumTT*sumSS));
	if (b != 0)
		znccValue = sumST / b;
	return znccValue;
}
extern "C"
float zncc2(float *d_img1, float *d_img2, long long int sx, long long int sy, long long int sz) {
	// d_img1, d_img2 value change after calculation
	float znccValue = -2.0;
	long long int totalSize = sx*sy*sz;
	double sumImg1 = 0, sumImg2 = 0, sumST = 0, sumSS = 0, sumTT = 0;
	float *h_img1 = (float *)malloc(totalSize * sizeof(float));
	sumImg1 = sum3Dgpu(d_img1, sx, sy, sz);
	sumImg2 = sum3Dgpu(d_img2, sx, sy, sz);
	addvaluegpu(d_img1, d_img1, -float(sumImg1) / float(totalSize), sx, sy, sz);
	addvaluegpu(d_img2, d_img2, -float(sumImg2) / float(totalSize), sx, sy, sz);
	hipMemcpy(h_img1, d_img1, totalSize * sizeof(float), hipMemcpyDeviceToHost);

	multi3Dgpu(d_img1, d_img1, d_img1, sx, sy, sz);
	sumTT = sum3Dgpu(d_img1, sx, sy, sz);
	hipMemcpy(d_img1, h_img1, totalSize * sizeof(float), hipMemcpyHostToDevice);
	multi3Dgpu(d_img1, d_img1, d_img2, sx, sy, sz);
	sumST = sum3Dgpu(d_img1, sx, sy, sz);
	multi3Dgpu(d_img2, d_img2, d_img2, sx, sy, sz);
	sumSS = sum3Dgpu(d_img2, sx, sy, sz);
	free(h_img1);
	float b = float(sqrt(sumTT*sumSS));
	if (b != 0)
		znccValue = sumST / b;
	return znccValue;
}
extern "C"

extern "C"
int reg3d_phasor0(long long int *shiftXYZ, float *h_img1, float *h_img2, long long int sx, long long int sy, long long int sz) {
	return 0;
}
extern "C"
int reg3d_phasor1(long long int *shiftXYZ, float *d_img1, float *d_img2, long long int sx, long long int sy, long long int sz) {
	int totalSize = sx * sy * sz;
	int totalSizeSpectrum = sz * sy*(sx / 2 + 1); // in complex floating format
	fComplex *d_Spectrum1 = NULL, *d_Spectrum2 = NULL;
	hipMalloc((void **)&d_Spectrum1, totalSizeSpectrum * sizeof(fComplex));
	hipMalloc((void **)&d_Spectrum2, totalSizeSpectrum * sizeof(fComplex));
	hipfftHandle
		fftPlanFwd,
		fftPlanInv;
	hipfftPlan3d(&fftPlanFwd, sz, sy, sx, HIPFFT_R2C);
	hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_img1, (hipfftComplex *)d_Spectrum2);
	conj3Dgpu(d_Spectrum1, d_Spectrum2, sz, sy, (sx / 2 + 1));
	hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_img2, (hipfftComplex *)d_Spectrum2);
	// multiplication and normalization
	multicomplexnorm3Dgpu(d_Spectrum2, d_Spectrum1, d_Spectrum2, sz, sy, (sx / 2 + 1));
	hipfftDestroy(fftPlanFwd);
	hipfftPlan3d(&fftPlanInv, sz, sy, sx, HIPFFT_C2R);
	float *d_phasor1 = (float *)d_Spectrum1;
	hipfftExecC2R(fftPlanInv, (hipfftComplex *)d_Spectrum2, (hipfftReal *)d_phasor1);
	hipfftDestroy(fftPlanInv);
	size_t corXYZ[3];
	float *d_phasor2 = (float *)d_Spectrum2;
	circshiftgpu(d_phasor2, d_phasor1, sx, sy, sz, round(sx / 2), round(sy / 2), round(sz / 2));
	float peakValue = max3Dgpu(&corXYZ[0], d_phasor2, sx, sy, sz);
	shiftXYZ[0] = long long int(corXYZ[0]) - sx / 2;
	shiftXYZ[1] = long long int(corXYZ[1]) - sy / 2;
	shiftXYZ[2] = long long int(corXYZ[2]) - sz / 2;
	hipFree(d_Spectrum1);
	hipFree(d_Spectrum2);
	
	// compare 8 cases based on cross-correlation
	long long int shiftX = shiftXYZ[0];
	long long int shiftY = shiftXYZ[1];
	long long int shiftZ = shiftXYZ[2];
	long long int xabs = abs(shiftX), yabs = abs(shiftY), zabs = abs(shiftZ);
	long long int beta = 4; // threshold value: only if shift is more than 1/beta of the image size
	if ((xabs >(sx /beta)) ||( yabs >(sy / beta)) || (zabs >(sz / beta))) {
		float *d_imgT = NULL, *d_crop1 = NULL, *d_crop2 = NULL;
		long long int sizex1, sizex2, sizey1, sizey2, sizez1, sizez2, sizex, sizey, sizez, sizeMaxCrop;
		sizex1 = xabs * sy * sz; sizex2 = (sx - xabs) * sy * sz;
		sizey1 = sx *yabs * sz; sizey2 = sx * (sy - yabs) * sz;
		sizez1 = sx * sy * zabs; sizez2 = sx * sy * (sz - zabs);
		sizex = (sizex1 > sizex2) ? sizex1 : sizex2;
		sizey = (sizey1 > sizey2) ? sizey1 : sizey2;
		sizez = (sizez1 > sizez2) ? sizez1 : sizez2;
		sizeMaxCrop = (sizex > sizey) ? sizex : sizey;
		sizeMaxCrop = (sizeMaxCrop > sizez) ? sizeMaxCrop : sizez;
		hipMalloc((void **)&d_imgT, totalSize * sizeof(float));
		hipMalloc((void **)&d_crop1, sizeMaxCrop * sizeof(float));
		hipMalloc((void **)&d_crop2, sizeMaxCrop * sizeof(float));
		circshiftgpu(d_imgT, d_img2, sx, sy, sz, -shiftX, -shiftY, -shiftZ);
		// encode the 8 cases as for loop
		long long int imSizeCropx[2], imSizeCropy[2], imSizeCropz[2];
		long long int imox[2], imoy[2], imoz[2];
		// index 0 records original shifts, index 1 switches the shift to the opposite case.  
		imSizeCropx[0] = sx - xabs; imSizeCropx[1] = xabs;
		if (shiftX > 0) {
			imox[0] = 0; imox[1] = sx - xabs;
		}
		else {
			imox[0] = xabs; imox[1] = 0;
		}
		imSizeCropy[0] = sy - yabs; imSizeCropy[1] = yabs;
		if (shiftY > 0) {
			imoy[0] = 0; imoy[1] = sy - yabs;
		}
		else {
			imoy[0] = yabs; imoy[1] = 0;
		}
		imSizeCropz[0] = sz - zabs; imSizeCropz[1] = zabs;
		if (shiftZ > 0) {
			imoz[0] = 0; imoz[1] = sz - zabs;
		}
		else {
			imoz[0] = zabs; imoz[1] = 0;
		}

		int indx = 0, indy = 0, indz = 0;
		float ccMax = -3, ccNow = 0;
		for (int i = 0; i < 2; i++) {
			if (imSizeCropx[i] > (sx / beta)) {
				for (int j = 0; j < 2; j++) {
					if (imSizeCropy[j] > (sy / beta)) {
						for (int k = 0; k < 2; k++) {
							if (imSizeCropz[k] > (sz / beta)) {
								cropgpu2(d_crop1, d_img1, imSizeCropx[i], imSizeCropy[j], imSizeCropz[k], sx, sy, sz, imox[i], imoy[j], imoz[k]);
								cropgpu2(d_crop2, d_imgT, imSizeCropx[i], imSizeCropy[j], imSizeCropz[k], sx, sy, sz, imox[i], imoy[j], imoz[k]);
								ccNow = zncc1(d_crop1, d_crop2, imSizeCropx[i], imSizeCropy[j], imSizeCropz[k]);
								if (ccMax < ccNow) {
									ccMax = ccNow;
									indx = i;
									indy = j;
									indz = k;
								}
							}
						}
					}
				}
			}
		}
		// if ind ==1, flip the coordinates
		if (indx == 1) {
			if (shiftX > 0)
				shiftXYZ[0] = shiftX - sx;
			else
				shiftXYZ[0] = shiftX + sx;
		}
		if (indy == 1) {
			if (shiftY > 0)
				shiftXYZ[1] = shiftY - sy;
			else
				shiftXYZ[1] = shiftY + sy;
		}
		if (indz == 1) {
			if (shiftZ > 0)
				shiftXYZ[2] = shiftZ - sz;
			else
				shiftXYZ[2] = shiftZ + sz;
		}
		hipFree(d_imgT);
		hipFree(d_crop1);
		hipFree(d_crop2);
	}
	return 0;
}
extern "C"
int reg3d_phasor2(long long int *shiftXYZ, float *h_img1, float *h_img2, long long int sx, long long int sy, long long int sz) {
	int totalSize = sx * sy * sz;
	int totalSizeSpectrum = sz * sy*(sx / 2 + 1); // in complex floating format
	fComplex *d_Spectrum1 = NULL, *d_Spectrum2 = NULL;
	hipMalloc((void **)&d_Spectrum1, totalSizeSpectrum * sizeof(fComplex));
	hipMalloc((void **)&d_Spectrum2, totalSizeSpectrum * sizeof(fComplex));
	float *d_img = (float *)d_Spectrum1;
	fComplex *h_Spectrum1 = (fComplex *)malloc(totalSizeSpectrum * sizeof(fComplex));
	hipfftHandle
		fftPlanFwd,
		fftPlanInv;
	hipfftPlan3d(&fftPlanFwd, sz, sy, sx, HIPFFT_R2C);
	hipMemcpy(d_img, h_img1, totalSize * sizeof(float), hipMemcpyHostToDevice);
	hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_img, (hipfftComplex *)d_Spectrum2);
	conj3Dgpu(d_Spectrum1, d_Spectrum2, sz, sy, (sx / 2 + 1));
	hipMemcpy(h_Spectrum1, d_Spectrum1, totalSizeSpectrum * sizeof(fComplex), hipMemcpyDeviceToHost);

	hipMemcpy(d_img, h_img2, totalSize * sizeof(float), hipMemcpyHostToDevice);
	hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_img, (hipfftComplex *)d_Spectrum2);
	// multiplication and normalization
	hipMemcpy(d_Spectrum1, h_Spectrum1, totalSizeSpectrum * sizeof(fComplex), hipMemcpyHostToDevice);
	multicomplexnorm3Dgpu(d_Spectrum2, d_Spectrum1, d_Spectrum2, sz, sy, (sx / 2 + 1));
	hipfftDestroy(fftPlanFwd);
	hipfftPlan3d(&fftPlanInv, sz, sy, sx, HIPFFT_C2R);
	hipfftExecC2R(fftPlanInv, (hipfftComplex *)d_Spectrum2, (hipfftReal *)d_img);
	hipfftDestroy(fftPlanInv);
	size_t corXYZ[3];
	float *d_phasor2 = (float *)d_Spectrum2;
	circshiftgpu(d_phasor2, d_img, sx, sy, sz, round(sx / 2), round(sy / 2), round(sz / 2));
	float peakValue = max3Dgpu(&corXYZ[0], d_phasor2, sx, sy, sz);
	shiftXYZ[0] = long long int(corXYZ[0]) - sx / 2;
	shiftXYZ[1] = long long int(corXYZ[1]) - sy / 2;
	shiftXYZ[2] = long long int(corXYZ[2]) - sz / 2;
	hipFree(d_Spectrum1);
	hipFree(d_Spectrum2);

	// compare 8 cases based on cross-correlation
	long long int shiftX = shiftXYZ[0];
	long long int shiftY = shiftXYZ[1];
	long long int shiftZ = shiftXYZ[2];
	long long int xabs = abs(shiftX), yabs = abs(shiftY), zabs = abs(shiftZ);
	long long int beta = 4; // threshold value: only if shift is more than 1/beta of the image size
	if ((xabs >(sx / beta)) || (yabs >(sy / beta)) || (zabs >(sz / beta))) {
		float *d_img1 = NULL, *d_imgT = NULL, *d_crop1 = NULL, *d_crop2 = NULL;
		long long int sizex1, sizex2, sizey1, sizey2, sizez1, sizez2, sizex, sizey, sizez, sizeMaxCrop;
		sizex1 = xabs * sy * sz; sizex2 = (sx - xabs) * sy * sz;
		sizey1 = sx *yabs * sz; sizey2 = sx * (sy - yabs) * sz;
		sizez1 = sx * sy * zabs; sizez2 = sx * sy * (sz - zabs);
		sizex = (sizex1 > sizex2) ? sizex1 : sizex2;
		sizey = (sizey1 > sizey2) ? sizey1 : sizey2;
		sizez = (sizez1 > sizez2) ? sizez1 : sizez2;
		sizeMaxCrop = (sizex > sizey) ? sizex : sizey;
		sizeMaxCrop = (sizeMaxCrop > sizez) ? sizeMaxCrop : sizez;
		hipMalloc((void **)&d_img1, totalSize * sizeof(float));
		hipMalloc((void **)&d_imgT, totalSize * sizeof(float));
		hipMalloc((void **)&d_crop1, sizeMaxCrop * sizeof(float));
		hipMalloc((void **)&d_crop2, sizeMaxCrop * sizeof(float));
		hipMemcpy(d_img1, h_img2, totalSize * sizeof(float), hipMemcpyHostToDevice);
		circshiftgpu(d_imgT, d_img1, sx, sy, sz, -shiftX, -shiftY, -shiftZ);
		hipMemcpy(d_img1, h_img1, totalSize * sizeof(float), hipMemcpyHostToDevice);
		// encode the 8 cases as for loop
		long long int imSizeCropx[2], imSizeCropy[2], imSizeCropz[2];
		long long int imox[2], imoy[2], imoz[2];
		// index 0 records original shifts, index 1 switches the shift to the opposite case.  
		imSizeCropx[0] = sx - xabs; imSizeCropx[1] = xabs;
		if (shiftX > 0) {
			imox[0] = 0; imox[1] = sx - xabs;
		}
		else {
			imox[0] = xabs; imox[1] = 0;
		}
		imSizeCropy[0] = sy - yabs; imSizeCropy[1] = yabs;
		if (shiftY > 0) {
			imoy[0] = 0; imoy[1] = sy - yabs;
		}
		else {
			imoy[0] = yabs; imoy[1] = 0;
		}
		imSizeCropz[0] = sz - zabs; imSizeCropz[1] = zabs;
		if (shiftZ > 0) {
			imoz[0] = 0; imoz[1] = sz - zabs;
		}
		else {
			imoz[0] = zabs; imoz[1] = 0;
		}

		int indx = 0, indy = 0, indz = 0;
		float ccMax = -3, ccNow = 0;
		for (int i = 0; i < 2; i++) {
			if (imSizeCropx[i] >(sx / beta)) {
				for (int j = 0; j < 2; j++) {
					if (imSizeCropy[j] >(sy / beta)) {
						for (int k = 0; k < 2; k++) {
							if (imSizeCropz[k] >(sz / beta)) {
								cropgpu2(d_crop1, d_img1, imSizeCropx[i], imSizeCropy[j], imSizeCropz[k], sx, sy, sz, imox[i], imoy[j], imoz[k]);
								cropgpu2(d_crop2, d_imgT, imSizeCropx[i], imSizeCropy[j], imSizeCropz[k], sx, sy, sz, imox[i], imoy[j], imoz[k]);
								ccNow = zncc1(d_crop1, d_crop2, imSizeCropx[i], imSizeCropy[j], imSizeCropz[k]);
								if (ccMax < ccNow) {
									ccMax = ccNow;
									indx = i;
									indy = j;
									indz = k;
								}
							}
						}
					}
				}
			}
		}
		// if ind ==1, flip the coordinates
		if (indx == 1) {
			if (shiftX > 0)
				shiftXYZ[0] = shiftX - sx;
			else
				shiftXYZ[0] = shiftX + sx;
		}
		if (indy == 1) {
			if (shiftY > 0)
				shiftXYZ[1] = shiftY - sy;
			else
				shiftXYZ[1] = shiftY + sy;
		}
		if (indz == 1) {
			if (shiftZ > 0)
				shiftXYZ[2] = shiftZ - sz;
			else
				shiftXYZ[2] = shiftZ + sz;
		}
		hipFree(d_img1);
		hipFree(d_imgT);
		hipFree(d_crop1);
		hipFree(d_crop2);
	}
	return 0;
}

int reg3d_affine0(float *h_reg, float *iTmx, float *h_img1, float *h_img2, long long int sx, long long int sy, long long int sz,
	int affMethod, bool flagTmx, float FTOL, int itLimit, bool verbose, float *records) {
	return 0;
}
extern "C"
int reg3d_affine1(float *d_reg, float *iTmx, float *d_img1, float *d_img2, long long int sx, long long int sy, long long int sz, 
	int affMethod, bool flagTmx, float FTOL, int itLimit, bool verbose, float *records) {
	// **** affine registration when GPU memory is sufficient: 3 images + 1 cuda array ***
	/*
	*** affine registration method: 
		0: no registration, transform d_img2 based on input matrix;
		1: translation only; 
		2: rigid body; 
		3: 7 degrees of freedom (translation, rotation, scaling equally in 3 dimensions)  
		4: 9 degrees of freedom(translation, rotation, scaling); 
		5: 12 degrees of freedom; 
		6: rigid body first, then do 12 degrees of freedom; 
		7: 3 DOF --> 6 DOF --> 9 DOF --> 12 DOF
	*** flagTmx: 
		true: use iTmx as input matrix;
		false: default; 
		
	*** records: 8 element array
		[1] -[3]: initial ZNCC (zero-normalized cross-correlation, negtive of the cost function), intermediate ZNCC, optimized ZNCC;
		[4] -[7]: single sub iteration time (in ms), total number of sub iterations, iteralation time (in s), whole registration time (in s);
	*/

	// ************get basic input images information ******************	
	// image size
	sxStatic1 = sx; syStatic1 = sy; szStatic1 = sz;
	sxStatic2 = sx; syStatic2 = sy; szStatic2 = sz;
	// total pixel count for each image
	long long int totalSize = sx*sy*sz;
	// ****************** Processing Starts*****************//
	// variables for memory and time cost records
	clock_t ctime0, ctime1, ctime2, ctime3, ctime4;
	ctime0 = clock();

	// *** no registration
	if (affMethod == 0) {
		if (flagTmx)
			(void)affinetrans3d1(d_reg, iTmx, d_img2, sx, sy, sz, sx, sy, sz);
		else {
			hipMemcpy(d_reg, d_img2, totalSize * sizeof(float), hipMemcpyDeviceToDevice);
			for (int j = 0; j < NDIM; j++) iTmx[j] = 0;
			iTmx[0] = iTmx[5] = iTmx[10] = 1;
		}
		ctime4 = clock();
		records[7] = (float)(ctime4 - ctime0) / CLOCKS_PER_SEC;
		if (verbose) {
			printf("...no registration performed!\n");
		}
		return 0;
	}
	// *** registration
	// for powell searching
	affCoef = (float *)malloc((NDIM) * sizeof(float));
	float *affCoefInitial = (float *)malloc((NDIM) * sizeof(float));
	static float *p = (float *)malloc((NDIM + 1) * sizeof(float));
	int iter;
	float fret, **xi;
	xi = matrix(1, NDIM, 1, NDIM);
	for (int i = 1; i <= NDIM; i++)
		for (int j = 1; j <= NDIM; j++)
			xi[i][j] = (i == j ? 1.0 : 0.0);
	for (int j = 0; j < NDIM; j++) affCoefInitial[j] = 0;
	affCoefInitial[0] = 1;
	affCoefInitial[5] = 1;
	affCoefInitial[10] = 1;

	float *affCoefTemp = (float *)malloc((NDIM) * sizeof(float));
	float **xi_dof9;
	static float *p_dof9 = (float *)malloc((10) * sizeof(float));
	xi_dof9 = matrix(1, 9, 1, 9);

	// **** allocate memory for the images: 
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipArray *d_Array;

	// *****************************************************
	// ************** Start processing ******************
	double
		sumImg1 = 0,
		sumImg2 = 0,
		sumSqr1 = 0;
	// ****** the definition of 12 DOF coefficients is totally diffrent with that of 3 DOF, 6 DOF, 7 DOF or 9 DOF;
	// if related to 3 DOF, 6 DOF, 7 DOF or 9 DOF (e.i. affMethod = 1, 2, 3, 4, 6, 7)
	// then perfrom initial affine transformation based on input matrix
	// *initialize transformation matrix
	if (flagTmx) {
		if (affMethod == 5) {
			// use input matrix as initialization if inputTmx is true
			memcpy(affCoefInitial, iTmx, NDIM * sizeof(float));
		}
		else {
			// make affine transformation
			(void)affinetrans3d1(d_reg, iTmx, d_img2, sx, sy, sz, sx, sy, sz);
		}
	}
	if(affMethod != 5) {
		xi_dof9 = matrix(1, 9, 1, 9);
		for (int i = 1; i <= 9; i++)
			for (int j = 1; j <= 9; j++)
				xi_dof9[i][j] = (i == j ? 1.0 : 0.0);
		p_dof9[0] = 0;
		p_dof9[1] = 0; p_dof9[2] = 0; p_dof9[3] = 0;
		p_dof9[4] = 0; p_dof9[5] = 0; p_dof9[6] = 0;
		p_dof9[7] = 1; p_dof9[8] = 1; p_dof9[9] = 1;
	}
	// *** preprocess source image
	if ((flagTmx)&&(affMethod != 5)) { // based on tranformed image
		sumImg2 = sum3Dgpu(d_reg, sx, sy, sz);
		addvaluegpu(d_reg, d_reg, -float(sumImg2) / float(totalSize), sx, sy, sz);
	}
	else {//based on input d_img2
		sumImg2 = sum3Dgpu(d_img2, sx, sy, sz);
		addvaluegpu(d_reg, d_img2, -float(sumImg2) / float(totalSize), sx, sy, sz);
	}
	// transfer source image into GPU array (later converted to texture memory)
	hipMalloc3DArray(&d_Array, &channelDesc, make_hipExtent(sx, sy, sz));
	cudaCheckErrors("****GPU memory allocating fails... GPU out of memory !!!!*****\n");
	cudacopydevicetoarray(d_Array, channelDesc, d_reg, sx, sy, sz);
	multi3Dgpu(d_reg, d_reg, d_reg, sx, sy, sz);
	sumSqr1 = sum3Dgpu(d_reg, sx, sy, sz);
	valueStatic = sqrt(sumSqr1);
	if (valueStatic == 0) {
		fprintf(stderr, "*** SD of image 2 is zero, empty image input or empty image after initial transformation **** \n");
		exit(1);
	}

	// *** preprocess target image
	sumImg1 = sum3Dgpu(d_img1, sx, sy, sz);
	addvaluegpu(d_reg, d_img1, -float(sumImg1) / float(totalSize), sx, sy, sz);
	multi3Dgpu(d_reg, d_reg, d_reg, sx, sy, sz);
	sumSqr1 = sum3Dgpu(d_reg, sx, sy, sz);
	valueStatic = sqrt(sumSqr1);
	if (valueStatic == 0) {
		fprintf(stderr, "*** SD of image 1 is zero, empty image input **** \n"); 
		exit(1);
	}
	addvaluegpu(d_reg, d_img1, -float(sumImg1) / float(totalSize), sx, sy, sz);
	cudaCheckErrors("****Image preprocessing fails...");

	// *** 3D registration begains
	// Create 3D texture for source image
	BindTexture(d_Array, channelDesc);
	// make target image as static
	d_imgStatic = d_reg;
	// calculate initial cost function value and time cost for each sub iteration
	ctime1 = clock();
	dof9Flag = false;
	matrix2p(affCoefInitial, p);
	ctime2 = clock();
	records[1] = -costfunc(p);
	records[4] = (float)(ctime2 - ctime1);
	if (verbose) {
		printf("...initial cross correlation value: %f;\n", records[1]);
		printf("...time cost for single sub iteration: %f ms;\n", records[4]);
	}

	itNumStatic = 0;
	switch (affMethod) {
	case 1:
		dof9Flag = true;
		dofNum = 3;
		powell(p_dof9, xi_dof9, dofNum, FTOL, &iter, &fret, costfunc, &itNumStatic, itLimit);
		break;
	case 2:
		dof9Flag = true;
		dofNum = 6;
		powell(p_dof9, xi_dof9, dofNum, FTOL, &iter, &fret, costfunc, &itNumStatic, itLimit);
		break;
	case 3:
		dof9Flag = true;
		dofNum = 7;
		powell(p_dof9, xi_dof9, dofNum, FTOL, &iter, &fret, costfunc, &itNumStatic, itLimit);
		break;
	case 4:
		dof9Flag = true;
		dofNum = 9;
		powell(p_dof9, xi_dof9, dofNum, FTOL, &iter, &fret, costfunc, &itNumStatic, itLimit);
		break;
	case 5:
		dof9Flag = false;
		dofNum = 12;
		powell(p, xi, dofNum, FTOL, &iter, &fret, costfunc, &itNumStatic, itLimit);
		break;
	case 6:
		// do 6 DOF --> 12 DOF
		dof9Flag = true;
		dofNum = 6;
		powell(p_dof9, xi_dof9, dofNum, 0.01, &iter, &fret, costfunc, &itNumStatic, itLimit);
		records[2] = -fret;
		if (verbose) {
			printf("...cross correlation value after 6 DOF: %f;\n", -fret);
		}
		// do DOF 12 registration
		dof9Flag = false;
		dofNum = 12;
		matrix2p(affCoef, p);
		powell(p, xi, dofNum, FTOL, &iter, &fret, costfunc, &itNumStatic, itLimit);
		break;
	case 7:
		// do 3 DOF --> 6 DOF --> 9 DOF --> 12 DOF
		dof9Flag = true;
		dofNum = 3;
		powell(p_dof9, xi_dof9, dofNum, 0.01, &iter, &fret, costfunc, &itNumStatic, itLimit);
		if (verbose) {
			printf("...cross correlation value after 3 DOF: %f;\n", -fret);
		}
		dofNum = 6;
		powell(p_dof9, xi_dof9, dofNum, 0.01, &iter, &fret, costfunc, &itNumStatic, itLimit);
		if (verbose) {
			printf("...cross correlation value after 6 DOF: %f;\n", -fret);
		}
		dofNum = 9;
		powell(p_dof9, xi_dof9, dofNum, 0.005, &iter, &fret, costfunc, &itNumStatic, itLimit);
		records[2] = -fret;
		if (verbose) {
			printf("...cross correlation value after 9 DOF: %f;\n", -fret);
		}
		// do DOF 12 registration
		dof9Flag = false;
		dofNum = 12;
		matrix2p(affCoef, p);
		powell(p, xi, dofNum, FTOL, &iter, &fret, costfunc, &itNumStatic, itLimit);
		break;
	default:
		printf("\n ****Wrong affine registration method is setup, no registraiton performed !!! **** \n");
	}
	if ((flagTmx) && (affMethod != 5)) {
		matrixmultiply(affCoefTemp, iTmx, affCoef); //final transformation matrix
		memcpy(affCoef, affCoefTemp, NDIM * sizeof(float));
	}
	UnbindTexture();
	memcpy(iTmx, affCoef, NDIM * sizeof(float));
	ctime3 = clock();
	records[3] = -fret; // negative of the mimized cost function value
	records[5] = (float)itNumStatic;
	records[6] = (float)(ctime3 - ctime2) / CLOCKS_PER_SEC;
	if (verbose) {
		printf("...optimized cross correlation value: %f;\n", records[3]);
		printf("...total sub iteration number: %d;\n", int(records[5]));
		printf("...time cost for all iterations: %f s;\n", records[6]);
	}
	// ****Perform affine transformation with optimized coefficients****//
	cudacopydevicetoarray(d_Array, channelDesc, d_img2, sx, sy, sz);
	BindTexture(d_Array, channelDesc);
	CopyTranMatrix(affCoef, NDIM * sizeof(float));
	affineTransform(d_reg, sx, sy, sz, sx, sy, sz);
	UnbindTexture();

	free(affCoefTemp);
	free(p_dof9);
	free_matrix(xi_dof9, 1, 9, 1, 9);

	free(affCoef);
	free(affCoefInitial);
	free(p);
	free_matrix(xi, 1, NDIM, 1, NDIM);

	//free GPU variables
	hipFreeArray(d_Array);
	ctime4 = clock();
	records[7] = (float)(ctime4 - ctime0) / CLOCKS_PER_SEC;
	if (verbose) {
		printf("...time cost for registration: %f s;\n", records[7]);
	}
	return 0;
}
extern "C"
int reg3d_affine2(float *d_reg, float *iTmx, float *h_img1, float *h_img2, long long int sx, long long int sy, long long int sz,
	int affMethod, bool flagTmx, float FTOL, int itLimit, bool verbose, float *records) {
	// **** affine registration when GPU memory is insufficient: 1 image + 1 cuda array ***
	/*
	*** affine registration method:
	0: no registration, transform d_img2 based on input matrix;
	1: translation only;
	2: rigid body;
	3: 7 degrees of freedom (translation, rotation, scaling equally in 3 dimensions)
	4: 9 degrees of freedom(translation, rotation, scaling);
	5: 12 degrees of freedom;
	6: rigid body first, then do 12 degrees of freedom;
	7: 3 DOF --> 6 DOF --> 9 DOF --> 12 DOF
	*** flagTmx:
	true: use iTmx as input matrix;
	false: default;

	*** records: 8 element array
	[1] -[3]: initial ZNCC (zero-normalized cross-correlation, negtive of the cost function), intermediate ZNCC, optimized ZNCC;
	[4] -[7]: single sub iteration time (in ms), total number of sub iterations, iteralation time (in s), whole registration time (in s);
	*/

	// ************get basic input images information ******************	
	// image size
	sxStatic1 = sx; syStatic1 = sy; szStatic1 = sz;
	sxStatic2 = sx; syStatic2 = sy; szStatic2 = sz;
	// total pixel count for each image
	long long int totalSize = sx*sy*sz;
	// ****************** Processing Starts*****************//
	// variables for memory and time cost records
	clock_t ctime0, ctime1, ctime2, ctime3, ctime4;
	ctime0 = clock();

	// *** no registration
	if (affMethod == 0) {
		if (flagTmx)
			(void)affinetrans3d2(d_reg, iTmx, h_img2, sx, sy, sz, sx, sy, sz);
		else {
			hipMemcpy(d_reg, h_img2, totalSize * sizeof(float), hipMemcpyHostToDevice);
			for (int j = 0; j < NDIM; j++) iTmx[j] = 0;
			iTmx[0] = iTmx[5] = iTmx[10] = 1;
		}
		ctime4 = clock();
		records[7] = (float)(ctime4 - ctime0) / CLOCKS_PER_SEC;
		if (verbose) {
			printf("...no registration performed!\n");
		}
		return 0;
	}
	// *** registration
	// for powell searching
	affCoef = (float *)malloc((NDIM) * sizeof(float));
	float *affCoefInitial = (float *)malloc((NDIM) * sizeof(float));
	static float *p = (float *)malloc((NDIM + 1) * sizeof(float));
	int iter;
	float fret, **xi;
	xi = matrix(1, NDIM, 1, NDIM);
	for (int i = 1; i <= NDIM; i++)
		for (int j = 1; j <= NDIM; j++)
			xi[i][j] = (i == j ? 1.0 : 0.0);
	for (int j = 0; j < NDIM; j++) affCoefInitial[j] = 0;
	affCoefInitial[0] = 1;
	affCoefInitial[5] = 1;
	affCoefInitial[10] = 1;

	float *affCoefTemp = (float *)malloc((NDIM) * sizeof(float));
	float **xi_dof9;
	static float *p_dof9 = (float *)malloc((10) * sizeof(float));
	xi_dof9 = matrix(1, 9, 1, 9);

	// **** allocate memory for the images: 
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipArray *d_Array;
	float *h_imgTemp = (float *)malloc(totalSize * sizeof(float));

	// *****************************************************
	// ************** Start processing ******************
	double
		sumImg1 = 0,
		sumImg2 = 0,
		sumSqr1 = 0;
	// ****** the definition of 12 DOF coefficients is totally diffrent with that of 3 DOF, 6 DOF, 7 DOF or 9 DOF;
	// if related to 3 DOF, 6 DOF, 7 DOF or 9 DOF (e.i. affMethod = 1, 2, 3, 4, 6, 7)
	// then perfrom initial affine transformation based on input matrix
	// *initialize transformation matrix
	if (flagTmx) {
		if (affMethod == 5) {
			// use input matrix as initialization if inputTmx is true
			memcpy(affCoefInitial, iTmx, NDIM * sizeof(float));
		}
		else {
			// make affine transformation
			(void)affinetrans3d2(d_reg, iTmx, h_img2, sx, sy, sz, sx, sy, sz);
		}
	}
	if (affMethod != 5) {
		xi_dof9 = matrix(1, 9, 1, 9);
		for (int i = 1; i <= 9; i++)
			for (int j = 1; j <= 9; j++)
				xi_dof9[i][j] = (i == j ? 1.0 : 0.0);
		p_dof9[0] = 0;
		p_dof9[1] = 0; p_dof9[2] = 0; p_dof9[3] = 0;
		p_dof9[4] = 0; p_dof9[5] = 0; p_dof9[6] = 0;
		p_dof9[7] = 1; p_dof9[8] = 1; p_dof9[9] = 1;
	}
	// *** preprocess source image
	if ((flagTmx) && (affMethod != 5)) { // based on tranformed image
		sumImg2 = sum3Dgpu(d_reg, sx, sy, sz);
		addvaluegpu(d_reg, d_reg, -float(sumImg2) / float(totalSize), sx, sy, sz);
	}
	else {//based on input d_img2
		hipMemcpy(d_reg, h_img2, totalSize * sizeof(float), hipMemcpyHostToDevice);
		sumImg2 = sum3Dgpu(d_reg, sx, sy, sz);
		addvaluegpu(d_reg, d_reg, -float(sumImg2) / float(totalSize), sx, sy, sz);
	}
	// transfer source image into GPU array (later converted to texture memory)
	hipMalloc3DArray(&d_Array, &channelDesc, make_hipExtent(sx, sy, sz));
	cudaCheckErrors("****GPU memory allocating fails... GPU out of memory !!!!*****\n");
	cudacopydevicetoarray(d_Array, channelDesc, d_reg, sx, sy, sz);
	multi3Dgpu(d_reg, d_reg, d_reg, sx, sy, sz);
	sumSqr1 = sum3Dgpu(d_reg, sx, sy, sz);
	valueStatic = sqrt(sumSqr1);
	if (valueStatic == 0) {
		fprintf(stderr, "*** SD of image 2 is zero, empty image input or empty image after initial transformation **** \n");
		exit(1);
	}

	// *** preprocess target image
	hipMemcpy(d_reg, h_img1, totalSize * sizeof(float), hipMemcpyHostToDevice);
	sumImg1 = sum3Dgpu(d_reg, sx, sy, sz);
	addvaluegpu(d_reg, d_reg, -float(sumImg1) / float(totalSize), sx, sy, sz);
	hipMemcpy(h_imgTemp, d_reg, totalSize * sizeof(float), hipMemcpyDeviceToHost);
	multi3Dgpu(d_reg, d_reg, d_reg, sx, sy, sz);
	sumSqr1 = sum3Dgpu(d_reg, sx, sy, sz);
	valueStatic = sqrt(sumSqr1);
	if (valueStatic == 0) {
		fprintf(stderr, "*** SD of image 1 is zero, empty image input **** \n");
		exit(1);
	}
	hipMemcpy(d_reg, h_imgTemp, totalSize * sizeof(float), hipMemcpyHostToDevice);
	cudaCheckErrors("****Image preprocessing fails...");

	// *** 3D registration begains
	// Create 3D texture for source image
	BindTexture(d_Array, channelDesc);
	// make target image as static
	d_imgStatic = d_reg;
	// calculate initial cost function value and time cost for each sub iteration
	ctime1 = clock();
	dof9Flag = false;
	matrix2p(affCoefInitial, p);
	ctime2 = clock();
	records[1] = -costfunc(p);
	records[4] = (float)(ctime2 - ctime1);
	if (verbose) {
		printf("...initial cross correlation value: %f;\n", records[1]);
		printf("...time cost for single sub iteration: %f ms;\n", records[4]);
	}

	itNumStatic = 0;
	switch (affMethod) {
	case 1:
		dof9Flag = true;
		dofNum = 3;
		powell(p_dof9, xi_dof9, dofNum, FTOL, &iter, &fret, costfunc, &itNumStatic, itLimit);
		break;
	case 2:
		dof9Flag = true;
		dofNum = 6;
		powell(p_dof9, xi_dof9, dofNum, FTOL, &iter, &fret, costfunc, &itNumStatic, itLimit);
		break;
	case 3:
		dof9Flag = true;
		dofNum = 7;
		powell(p_dof9, xi_dof9, dofNum, FTOL, &iter, &fret, costfunc, &itNumStatic, itLimit);
		break;
	case 4:
		dof9Flag = true;
		dofNum = 9;
		powell(p_dof9, xi_dof9, dofNum, FTOL, &iter, &fret, costfunc, &itNumStatic, itLimit);
		break;
	case 5:
		dof9Flag = false;
		dofNum = 12;
		powell(p, xi, dofNum, FTOL, &iter, &fret, costfunc, &itNumStatic, itLimit);
		break;
	case 6:
		// do 6 DOF --> 12 DOF
		dof9Flag = true;
		dofNum = 6;
		powell(p_dof9, xi_dof9, dofNum, 0.01, &iter, &fret, costfunc, &itNumStatic, itLimit);
		records[2] = -fret;
		if (verbose) {
			printf("...cross correlation value after 6 DOF: %f;\n", -fret);
		}
		// do DOF 12 registration
		dof9Flag = false;
		dofNum = 12;
		matrix2p(affCoef, p);
		powell(p, xi, dofNum, FTOL, &iter, &fret, costfunc, &itNumStatic, itLimit);
		break;
	case 7:
		// do 3 DOF --> 6 DOF --> 9 DOF --> 12 DOF
		dof9Flag = true;
		dofNum = 3;
		powell(p_dof9, xi_dof9, dofNum, 0.01, &iter, &fret, costfunc, &itNumStatic, itLimit);
		if (verbose) {
			printf("...cross correlation value after 3 DOF: %f;\n", -fret);
		}
		dofNum = 6;
		powell(p_dof9, xi_dof9, dofNum, 0.01, &iter, &fret, costfunc, &itNumStatic, itLimit);
		if (verbose) {
			printf("...cross correlation value after 6 DOF: %f;\n", -fret);
		}
		dofNum = 9;
		powell(p_dof9, xi_dof9, dofNum, 0.005, &iter, &fret, costfunc, &itNumStatic, itLimit);
		records[2] = -fret;
		if (verbose) {
			printf("...cross correlation value after 9 DOF: %f;\n", -fret);
		}
		// do DOF 12 registration
		dof9Flag = false;
		dofNum = 12;
		matrix2p(affCoef, p);
		powell(p, xi, dofNum, FTOL, &iter, &fret, costfunc, &itNumStatic, itLimit);
		break;
	default:
		printf("\n ****Wrong affine registration method is setup, no registraiton performed !!! **** \n");
	}
	if ((flagTmx) && (affMethod != 5)) {
		matrixmultiply(affCoefTemp, iTmx, affCoef); //final transformation matrix
		memcpy(affCoef, affCoefTemp, NDIM * sizeof(float));
	}
	UnbindTexture();
	memcpy(iTmx, affCoef, NDIM * sizeof(float));
	ctime3 = clock();
	records[3] = -fret; // negative of the mimized cost function value
	records[5] = (float)itNumStatic;
	records[6] = (float)(ctime3 - ctime2) / CLOCKS_PER_SEC;
	if (verbose) {
		printf("...optimized cross correlation value: %f;\n", records[3]);
		printf("...total sub iteration number: %d;\n", int(records[5]));
		printf("...time cost for all iterations: %f s;\n", records[6]);
	}
	// ****Perform affine transformation with optimized coefficients****//
	cudacopyhosttoarray(d_Array, channelDesc, h_img2, sx, sy, sz);
	BindTexture(d_Array, channelDesc);
	CopyTranMatrix(affCoef, NDIM * sizeof(float));
	affineTransform(d_reg, sx, sy, sz, sx, sy, sz);
	UnbindTexture();
	
	free(h_imgTemp);

	free(affCoefTemp);
	free(p_dof9);
	free_matrix(xi_dof9, 1, 9, 1, 9);

	free(affCoef);
	free(affCoefInitial);
	free(p);
	free_matrix(xi, 1, NDIM, 1, NDIM);

	//free GPU variables
	hipFreeArray(d_Array);
	ctime4 = clock();
	records[7] = (float)(ctime4 - ctime0) / CLOCKS_PER_SEC;
	if (verbose) {
		printf("...time cost for registration: %f s;\n", records[7]);
	}
	return 0;
}

// Deconvolution
extern "C"
void genOTFgpu(fComplex *d_odata, float *d_idata, long long int sx, long long int sy, long long int sz, long long int sx2,
	long long int sy2, long long int sz2, bool normFlag) {
	long long int totalSizeIn = sx2 * sy2 * sz2;
	long long int totalSizeOut = sx * sy * sz;
	long long int totalSizeMax = (totalSizeIn > totalSizeOut)?totalSizeIn:totalSizeOut;
	float *d_temp = NULL;
	cudaStatus = hipMalloc((void **)&d_temp, totalSizeMax * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "*** FAILED - ABORTING: GPU memory allocating error when calculating OTF \n");
		exit(1);
	}

	if (normFlag) {
		double sumValue = sum3Dgpu(d_idata, sx2, sy2, sz2);
		multivaluegpu(d_temp, d_idata, (float)(1 / sumValue), sx2, sy2, sz2);
	}
	else
		hipMemcpy(d_temp, d_idata, totalSizeIn * sizeof(float), hipMemcpyDeviceToDevice);
	if ((sx<sx2) || (sy<sy2) || (sz<sz2)) {
		alignsize3Dgpu((float *)d_odata, d_temp, sx, sy, sz, sx2, sy2, sz2);
		padPSFgpu(d_temp, (float *)d_odata, sx, sy, sz, sx, sy, sz);
	}
	else {
		padPSFgpu((float *)d_odata, d_temp, sx, sy, sz, sx2, sy2, sz2);
		hipMemcpy(d_temp, d_odata, totalSizeOut * sizeof(float), hipMemcpyDeviceToDevice);
	}	
	hipfftHandle
		fftPlanFwd;
	hipfftPlan3d(&fftPlanFwd, sx, sy, sz, HIPFFT_R2C);
	hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_temp, (hipfftComplex *)d_odata);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "*** FAILED - ABORTING: cufftPlan error when calculating OTF \n");
		exit(1);
	}
	hipFree(d_temp);
	hipfftDestroy(fftPlanFwd);
}

extern "C"
int decon_singleview_OTF0(float *h_decon, float *h_img, fftwf_complex *h_OTF, fftwf_complex *h_OTF_bp,
	long long int sx, long long int sy, long long int sz, int itNumForDecon, bool flagConstInitial) {
	// **** single view deconvolution with OTF interface on CPU ***
	// image size
	long long int totalSize = sx*sy*sz; // in floating format
	long long int totalSizeSpectrum = sx * sy*(sz / 2 + 1); // in complex floating format
	clock_t start, end;
	start = clock();
	float *h_StackA = h_img, *h_StackE = h_decon;
	float *h_StackT = (float *)malloc(totalSize * sizeof(float));
	fftwf_complex *h_StackESpectrum = (fftwf_complex *)malloc(totalSizeSpectrum * sizeof(fftwf_complex));

	// initialize estimation
	maxvaluecpu(h_StackA, h_StackA, (float)(SMALLVALUE), totalSize);
	// initialize estimation
	if (flagConstInitial) { // use constant mean value as initial
		float meanValue = (float)sumcpu(h_StackA, totalSize);
		memset(h_StackE, 0, totalSize * sizeof(float));
		addvaluecpu(h_StackE, h_StackE, meanValue, totalSize);
	}
	else { // use measured images as initial
		memcpy(h_StackE, h_StackA, totalSize * sizeof(float));
	}

	fftwf_plan stackE2Spectrum = fftwf_plan_dft_r2c_3d(sx, sy, sz, h_StackE, h_StackESpectrum, FFTW_MEASURE);
	fftwf_plan stackT2Spectrum = fftwf_plan_dft_r2c_3d(sx, sy, sz, h_StackT, h_StackESpectrum, FFTW_MEASURE);
	fftwf_plan spectrum2StackT = fftwf_plan_dft_c2r_3d(sx, sy, sz, h_StackESpectrum, h_StackT, FFTW_MEASURE);
	printf("...Start CPU Decon\n");
	for (int itNum = 1; itNum <= itNumForDecon; itNum++) {
		fftwf_execute(stackE2Spectrum);
		multicomplexcpu((fComplex *)h_StackESpectrum, (fComplex *)h_StackESpectrum, (fComplex *)h_OTF, sx * sy * (sz / 2 + 1));
		fftwf_execute(spectrum2StackT);

		divcpu(h_StackT, h_StackA, h_StackT, totalSize);

		fftwf_execute(stackT2Spectrum);
		multicomplexcpu((fComplex *)h_StackESpectrum, (fComplex *)h_StackESpectrum, (fComplex *)h_OTF_bp, sx * sy * (sz / 2 + 1));
		fftwf_execute(spectrum2StackT);
		multicpu(h_StackE, h_StackE, h_StackT, totalSize);//
	}
	free(h_StackT);
	free(h_StackESpectrum);
	fftwf_destroy_plan(stackE2Spectrum);
	fftwf_destroy_plan(stackT2Spectrum);
	fftwf_destroy_plan(spectrum2StackT);
	end = clock();
	printf("...Time cost for decon is %2.3f s\n", (float)(end - start) / CLOCKS_PER_SEC);
	return 0;
}
extern "C"
int decon_singleview_OTF1(float *d_decon, float *d_img, fComplex *d_OTF, fComplex *d_OTF_bp, 
	long long int sx, long long int sy, long long int sz, int itNumForDecon, bool flagConstInitial) {
	// **** single view deconvolution with OTF interface when GPU memory is sufficient ***
	// image size
	long long int totalSize = sx*sy*sz; // in floating format
	long long int totalSizeSpectrum = sx * sy*(sz / 2 + 1); // in complex floating format
	size_t freeMem = 0, totalMem = 0;
	hipfftHandle
		fftPlanFwd,
		fftPlanInv;
	clock_t start, end;
	start = clock();
	float *d_StackA = d_img, *d_StackE = d_decon;
	float *d_StackT = NULL;
	fComplex *d_StackESpectrum = NULL;
	hipMalloc((void **)&d_StackT, totalSize * sizeof(float));
	hipMalloc((void **)&d_StackESpectrum, totalSizeSpectrum * sizeof(fComplex));

	// initialize estimation
	maxvalue3Dgpu(d_StackA, d_StackA, (float)(SMALLVALUE), sx, sy, sz);
	if(flagConstInitial) {// use constant mean value as initial
		float meanValue = (float)sum3Dgpu(d_StackA, sx, sy, sz);
		hipMemset(d_StackE, 0, totalSize * sizeof(float));
		addvaluegpu(d_StackE, d_StackE, meanValue, sx, sy, sz);
	}
	else { // use measured image as initial
		hipMemcpy(d_StackE, d_StackA, totalSize * sizeof(float), hipMemcpyDeviceToDevice);
	}
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "*** FAILED - ABORTING: initial image preparation failed \n");
		exit(1);
	}
	// Create FFT plans
	hipfftPlan3d(&fftPlanFwd, sx, sy, sz, HIPFFT_R2C);
	hipfftPlan3d(&fftPlanInv, sx, sy, sz, HIPFFT_C2R);
	hipMemGetInfo(&freeMem, &totalMem);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "*** FAILED - ABORTING: cufftPlan error \n");
		exit(1);
	}
	printf("...GPU free memory (before decon iteration) is %.0f MBites\n", (float)freeMem / 1048576.0f);
	for (int itNum = 1; itNum <= itNumForDecon; itNum++) {
		// forward
		hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_StackE, (hipfftComplex *)d_StackESpectrum);
		multicomplex3Dgpu(d_StackESpectrum, d_StackESpectrum, d_OTF, sx, sy, (sz / 2 + 1));
		hipfftExecC2R(fftPlanInv, (hipfftComplex *)d_StackESpectrum, (hipfftReal *)d_StackT);
		div3Dgpu(d_StackT, d_StackA, d_StackT, sx, sy, sz); 
		// backward
		hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_StackT, (hipfftComplex *)d_StackESpectrum);
		multicomplex3Dgpu(d_StackESpectrum, d_StackESpectrum, d_OTF_bp, sx, sy, (sz / 2 + 1));
		hipfftExecC2R(fftPlanInv, (hipfftComplex *)d_StackESpectrum, (hipfftReal *)d_StackT);																		  
		multi3Dgpu(d_StackE, d_StackE, d_StackT, sx, sy, sz);
		maxvalue3Dgpu(d_StackE, d_StackE, float(SMALLVALUE), sx, sy, sz); // eliminate possible negative values
	}
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "*** FAILED - ABORTING: decon iterration error \n");
		exit(1);
	}
	hipFree(d_StackT); hipFree(d_StackESpectrum);
	hipfftDestroy(fftPlanFwd);
	hipfftDestroy(fftPlanInv);
	hipMemGetInfo(&freeMem, &totalMem);
	printf("...GPU free memory (after decon iteration) is %.0f MBites\n", (float)freeMem / 1048576.0f);
	end = clock();
	printf("...Time cost for decon is %2.3f s\n", (float)(end - start) / CLOCKS_PER_SEC);
	return 0;
}
extern "C"
int decon_singleview_OTF2(float *d_decon, float *d_img, fComplex *h_OTF, fComplex *h_OTF_bp,
	long long int sx, long long int sy, long long int sz, int itNumForDecon, bool flagConstInitial) {
	// **** single view deconvolution with OTF interface when GPU memory is insufficient: 2 images + 2 fftPlans ***
	// **** d_decon and d_img should have total size: sx * sy*(sz / 2 + 1) * sizeof(float) to store image spectrum
	// image size
	long long int totalSize = sx*sy*sz; // in floating format
	long long int totalSizeSpectrum = sx * sy*(sz / 2 + 1); // in complex floating format
	// *****
	size_t freeMem = 0, totalMem = 0;
	hipfftHandle
		fftPlanFwd,
		fftPlanInv;
	clock_t start, end;
	start = clock();
	float *h_StackA = NULL, *h_StackE = NULL;
	h_StackA = (float *)malloc(totalSize * sizeof(float));
	h_StackE = (float *)malloc(totalSize * sizeof(float));


	float *d_StackA = d_img, *d_StackE = d_decon;
	fComplex  *d_OTF = NULL, *d_OTF_bp = NULL, *d_StackESpectrum = NULL;
	// initialize estimation
	maxvalue3Dgpu(d_StackA, d_StackA, (float)(SMALLVALUE), sx, sy, sz);
	hipMemcpy(h_StackA, d_StackA, totalSize * sizeof(float), hipMemcpyDeviceToHost);
	//if (initialFlag) // use measured image as initial
	if (flagConstInitial) { // use constant mean value as initial
		float meanValue = (float)sum3Dgpu(d_StackA, sx, sy, sz);	
		hipMemset(d_StackA, 0, totalSize * sizeof(float));
		addvaluegpu(d_StackA, d_StackA, meanValue, sx, sy, sz);
	}
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "*** FAILED - ABORTING: initial image preparation failed \n");
		exit(1);
	}
	hipMemcpy(h_StackE, d_StackA, totalSize * sizeof(float), hipMemcpyDeviceToHost);
	d_OTF = (fComplex *)d_StackA; // share the same physic memory
	d_OTF_bp = (fComplex *)d_StackA; // share the same physic memory
	d_StackESpectrum = (fComplex *)d_StackE; // share the same physic memory

	// Create FFT plans
	hipfftPlan3d(&fftPlanFwd, sx, sy, sz, HIPFFT_R2C);
	hipfftPlan3d(&fftPlanInv, sx, sy, sz, HIPFFT_C2R);
	hipMemGetInfo(&freeMem, &totalMem);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "*** FAILED - ABORTING: cufftPlan error \n");
		exit(1);
	}
	printf("...GPU free memory (before decon iteration) is %.0f MBites\n", (float)freeMem / 1048576.0f);

	for (int itNum = 1; itNum <= itNumForDecon; itNum++) {
		// forward
		hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_StackA, (hipfftComplex *)d_StackESpectrum);
		hipMemcpy(d_OTF, h_OTF, totalSizeSpectrum * sizeof(fComplex), hipMemcpyHostToDevice);
		multicomplex3Dgpu(d_StackESpectrum, d_StackESpectrum, d_OTF, sx, sy, (sz / 2 + 1));
		hipfftExecC2R(fftPlanInv, (hipfftComplex *)d_StackESpectrum, (hipfftReal *)d_StackA);
		hipMemcpy(d_StackE, h_StackA, totalSize * sizeof(float), hipMemcpyHostToDevice);
		div3Dgpu(d_StackA, d_StackE, d_StackA, sx, sy, sz); 

		// backward
		hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_StackA, (hipfftComplex *)d_StackESpectrum);
		hipMemcpy(d_OTF_bp, h_OTF_bp, totalSizeSpectrum * sizeof(fComplex), hipMemcpyHostToDevice);
		multicomplex3Dgpu(d_StackESpectrum, d_StackESpectrum, d_OTF_bp, sx, sy, (sz / 2 + 1));
		hipfftExecC2R(fftPlanInv, (hipfftComplex *)d_StackESpectrum, (hipfftReal *)d_StackA);																			  
		hipMemcpy(d_StackE, h_StackE, totalSize * sizeof(float), hipMemcpyHostToDevice);
		multi3Dgpu(d_StackA, d_StackE, d_StackA, sx, sy, sz);
		maxvalue3Dgpu(d_StackA, d_StackA, float(SMALLVALUE), sx, sy, sz); // eliminate possible negative values
		hipMemcpy(h_StackE, d_StackA, totalSize * sizeof(float), hipMemcpyDeviceToHost);
	}
	hipMemcpy(d_StackE, d_StackA, totalSize * sizeof(float), hipMemcpyDeviceToDevice);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "*** FAILED - ABORTING: decon iterration error \n");
		exit(1);
	}
	free(h_StackA); free(h_StackE);
	hipfftDestroy(fftPlanFwd);
	hipfftDestroy(fftPlanInv);
	hipMemGetInfo(&freeMem, &totalMem);
	printf("...GPU free memory (after decon iteration) is %.0f MBites\n", (float)freeMem / 1048576.0f);
	end = clock();
	printf("...Time cost for decon is %2.3f s\n", (float)(end - start) / CLOCKS_PER_SEC);
	return 0;
}

extern "C"
int decon_dualview_OTF0(float *h_decon, float *h_img1, float *h_img2, fftwf_complex *h_OTF1, fftwf_complex *h_OTF2, fftwf_complex *h_OTF_bp1,
	fftwf_complex *h_OTF_bp2, long long int sx, long long int sy, long long int sz, int itNumForDecon, bool flagConstInitial) {
	// **** dual-view deconvolution with OTF interface on CPU ***
	// image size
	long long int totalSize = sx*sy*sz; // in floating format
	long long int totalSizeSpectrum = sx * sy*(sz / 2 + 1); // in complex floating format
	clock_t start, end;
	start = clock();
	float *h_StackA = h_img1, *h_StackB = h_img2, *h_StackE = h_decon;
	float *h_StackT = (float *)malloc(totalSize * sizeof(float));
	fftwf_complex *h_StackESpectrum = (fftwf_complex *)malloc(totalSizeSpectrum * sizeof(fftwf_complex));

	// initialize estimation
	maxvaluecpu(h_StackA, h_StackA, (float)(SMALLVALUE), totalSize);
	maxvaluecpu(h_StackB, h_StackB, (float)(SMALLVALUE), totalSize);
	// initialize estimation
	if (flagConstInitial) { // use constant mean value as initial
		float meanValue1 = (float)sumcpu(h_StackA, totalSize);
		float meanValue2 = (float)sumcpu(h_StackB, totalSize);
		memset(h_StackE, 0, totalSize * sizeof(float));
		addvaluecpu(h_StackE, h_StackE, (meanValue1 + meanValue2) / 2, totalSize);
	}
	else { // use measured images as initial
		addcpu(h_StackE, h_StackA, h_StackB, totalSize);
		multivaluecpu(h_StackE, h_StackE, (float)0.5, totalSize);
	}

	fftwf_plan stackE2Spectrum = fftwf_plan_dft_r2c_3d(sx, sy, sz, h_StackE, h_StackESpectrum, FFTW_MEASURE);
	fftwf_plan stackT2Spectrum = fftwf_plan_dft_r2c_3d(sx, sy, sz, h_StackT, h_StackESpectrum, FFTW_MEASURE);
	fftwf_plan spectrum2StackT = fftwf_plan_dft_c2r_3d(sx, sy, sz, h_StackESpectrum, h_StackT, FFTW_MEASURE);
	printf("...Start CPU Decon\n");
	for (int itNum = 1; itNum <= itNumForDecon; itNum++) {
		fftwf_execute(stackE2Spectrum);
		multicomplexcpu((fComplex *)h_StackESpectrum, (fComplex *)h_StackESpectrum, (fComplex *)h_OTF1, sx * sy * (sz / 2 + 1));
		fftwf_execute(spectrum2StackT);
		//printf("here!\n");

		divcpu(h_StackT, h_StackA, h_StackT, totalSize);

		fftwf_execute(stackT2Spectrum);
		multicomplexcpu((fComplex *)h_StackESpectrum, (fComplex *)h_StackESpectrum, (fComplex *)h_OTF_bp1, sx * sy * (sz / 2 + 1));
		fftwf_execute(spectrum2StackT);
		multicpu(h_StackE, h_StackE, h_StackT, totalSize);//
		return 0;
		fftwf_execute(stackE2Spectrum);
		multicomplexcpu((fComplex *)h_StackESpectrum, (fComplex *)h_StackESpectrum, (fComplex *)h_OTF2, sx * sy * (sz / 2 + 1));
		fftwf_execute(spectrum2StackT);

		divcpu(h_StackT, h_StackB, h_StackT, totalSize);

		fftwf_execute(stackT2Spectrum);
		multicomplexcpu((fComplex *)h_StackESpectrum, (fComplex *)h_StackESpectrum, (fComplex *)h_OTF_bp2, sx * sy * (sz / 2 + 1));
		fftwf_execute(spectrum2StackT);
		multicpu(h_StackE, h_StackE, h_StackT, totalSize);//
	}
	free(h_StackT);
	free(h_StackESpectrum);
	fftwf_destroy_plan(stackE2Spectrum);
	fftwf_destroy_plan(stackT2Spectrum);
	fftwf_destroy_plan(spectrum2StackT);
	end = clock();
	printf("...Time cost for decon is %2.3f s\n", (float)(end - start) / CLOCKS_PER_SEC);

	return 0;
}
extern "C"
int decon_dualview_OTF1(float *d_decon, float *d_img1, float *d_img2, fComplex *d_OTF1, fComplex *d_OTF2, fComplex *d_OTF_bp1,
	fComplex *d_OTF_bp2, long long int sx, long long int sy, long long int sz, int itNumForDecon, bool flagConstInitial) {
	// **** dual-view deconvolution with OTF interface when GPU memory is sufficient: 9 images + 2 fftPlans ***
	// image size
	long long int totalSize = sx*sy*sz; // in floating format
	long long int totalSizeSpectrum = sx * sy*(sz / 2 + 1); // in complex floating format
	size_t freeMem = 0, totalMem = 0;
	hipfftHandle
		fftPlanFwd,
		fftPlanInv;
	clock_t start, end;
	start = clock();
	float *d_StackA = d_img1, *d_StackB = d_img2, *d_StackE = d_decon;
	float *d_StackT = NULL;
	fComplex *d_StackESpectrum = NULL;
	hipMalloc((void **)&d_StackT, totalSize * sizeof(float));
	hipMalloc((void **)&d_StackESpectrum, totalSizeSpectrum * sizeof(fComplex));

	// initialize estimation
	maxvalue3Dgpu(d_StackA, d_StackA, (float)(SMALLVALUE), sx, sy, sz);
	maxvalue3Dgpu(d_StackB, d_StackB, (float)(SMALLVALUE), sx, sy, sz);
	// initialize estimation
	if (flagConstInitial) { // use constant mean value as initial
		float meanValue1 = (float)sum3Dgpu(d_StackA, sx, sy, sz);
		float meanValue2 = (float)sum3Dgpu(d_StackB, sx, sy, sz);
		hipMemset(d_StackE, 0, totalSize * sizeof(float));
		addvaluegpu(d_StackE, d_StackE, (meanValue1 + meanValue2) / 2, sx, sy, sz);
	}
	else { // use measured images as initial
		add3Dgpu(d_StackE, d_StackA, d_StackB, sx, sy, sz);
		multivaluegpu(d_StackE, d_StackE, (float)0.5, sx, sy, sz); 
	}
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "*** FAILED - ABORTING: initial image preparation failed \n");
		exit(1);
	}
	// Create FFT plans
	hipfftPlan3d(&fftPlanFwd, sx, sy, sz, HIPFFT_R2C);
	hipfftPlan3d(&fftPlanInv, sx, sy, sz, HIPFFT_C2R);
	hipMemGetInfo(&freeMem, &totalMem);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "*** FAILED - ABORTING: cufftPlan error \n");
		exit(1);
	}
	printf("...GPU free memory (before decon iteration) is %.0f MBites\n", (float)freeMem / 1048576.0f);
	for (int itNum = 1; itNum <= itNumForDecon; itNum++) {
		// ### 1st view
		hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_StackE, (hipfftComplex *)d_StackESpectrum);
		multicomplex3Dgpu(d_StackESpectrum, d_StackESpectrum, d_OTF1, sx, sy, (sz / 2 + 1));
		hipfftExecC2R(fftPlanInv, (hipfftComplex *)d_StackESpectrum, (hipfftReal *)d_StackT);
		div3Dgpu(d_StackT, d_StackA, d_StackT, sx, sy, sz);   
															
		hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_StackT, (hipfftComplex *)d_StackESpectrum);
		multicomplex3Dgpu(d_StackESpectrum, d_StackESpectrum, d_OTF_bp1, sx, sy, (sz / 2 + 1));
		hipfftExecC2R(fftPlanInv, (hipfftComplex *)d_StackESpectrum, (hipfftReal *)d_StackT);
																						  
		multi3Dgpu(d_StackE, d_StackE, d_StackT, sx, sy, sz);//
		maxvalue3Dgpu(d_StackE, d_StackE, float(SMALLVALUE), sx, sy, sz);

		// ### 2nd view
		hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_StackE, (hipfftComplex *)d_StackESpectrum);//
		multicomplex3Dgpu(d_StackESpectrum, d_StackESpectrum, d_OTF2, sx, sy, (sz / 2 + 1));
		hipfftExecC2R(fftPlanInv, (hipfftComplex *)d_StackESpectrum, (hipfftReal *)d_StackT);
		div3Dgpu(d_StackT, d_StackB, d_StackT, sx, sy, sz);//
																
		hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_StackT, (hipfftComplex *)d_StackESpectrum);
		multicomplex3Dgpu(d_StackESpectrum, d_StackESpectrum, d_OTF_bp2, sx, sy, (sz / 2 + 1));
		hipfftExecC2R(fftPlanInv, (hipfftComplex *)d_StackESpectrum, (hipfftReal *)d_StackT);

		multi3Dgpu(d_StackE, d_StackE, d_StackT, sx, sy, sz);
		maxvalue3Dgpu(d_StackE, d_StackE, float(SMALLVALUE), sx, sy, sz);
	}
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "*** FAILED - ABORTING: decon iterration error \n");
		exit(1);
	}
	hipFree(d_StackT); hipFree(d_StackESpectrum);
	hipfftDestroy(fftPlanFwd);
	hipfftDestroy(fftPlanInv);
	hipMemGetInfo(&freeMem, &totalMem);
	printf("...GPU free memory (after decon iteration) is %.0f MBites\n", (float)freeMem / 1048576.0f);
	end = clock();
	printf("...Time cost for decon is %2.3f s\n", (float)(end - start) / CLOCKS_PER_SEC);
	return 0;
}
extern "C"
int decon_dualview_OTF2(float *d_decon, float *d_img1, float *h_img2, fComplex *h_OTF1, fComplex *h_OTF2, fComplex *h_OTF_bp1,
	fComplex *h_OTF_bp2, long long int sx, long long int sy, long long int sz, int itNumForDecon, bool flagConstInitial) {
	// **** dual-view deconvolution with OTF interface when GPU memory is insufficient: 2 images + 2 fftPlans ***
	// **** d_decon and d_img should have total size: sx * sy*(sz / 2 + 1) * sizeof(float) to store image spectrum
	// image size
	long long int totalSize = sx*sy*sz; // in floating format
	long long int totalSizeSpectrum = sx * sy*(sz / 2 + 1); // in complex floating format
															// *****
	size_t freeMem = 0, totalMem = 0;
	hipfftHandle
		fftPlanFwd,
		fftPlanInv;
	clock_t start, end;
	start = clock();
	float *h_StackA = NULL, *h_StackB = NULL, *h_StackE = NULL;
	h_StackA = (float *)malloc(totalSize * sizeof(float));
	h_StackB = (float *)malloc(totalSize * sizeof(float));
	h_StackE = (float *)malloc(totalSize * sizeof(float));

	float *d_StackA = d_img1, *d_StackE = d_decon;
	float *d_StackB = NULL;
	fComplex  *d_OTF = NULL, *d_StackESpectrum = NULL;
	d_StackESpectrum = (fComplex *)d_StackA;
	d_OTF = (fComplex *)d_StackE;
	cudaStatus = hipGetLastError();

	// initialize estimation
	hipMalloc((void **)&d_StackB, totalSize * sizeof(float));
	maxvalue3Dgpu(d_StackA, d_StackA, (float)(SMALLVALUE), sx, sy, sz);
	hipMemcpy(d_StackB, h_img2, totalSize * sizeof(float), hipMemcpyHostToDevice);
	maxvalue3Dgpu(d_StackB, d_StackB, (float)(SMALLVALUE), sx, sy, sz);
	hipMemcpy(h_StackA, d_StackA, totalSize * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_StackB, d_StackB, totalSize * sizeof(float), hipMemcpyDeviceToHost);
	if (flagConstInitial) { // use constant mean value as initial
		float meanValue1 = (float)sum3Dgpu(d_StackA, sx, sy, sz);
		float meanValue2 = (float)sum3Dgpu(d_StackB, sx, sy, sz);
		hipMemset(d_StackE, 0, totalSize * sizeof(float));
		addvaluegpu(d_StackE, d_StackE, (meanValue1 + meanValue2) / 2, sx, sy, sz);
	}
	else { // use measured images as initial
		add3Dgpu(d_StackE, d_StackA, d_StackB, sx, sy, sz);
		multivaluegpu(d_StackE, d_StackE, (float)0.5, sx, sy, sz);
	}
	hipMemcpy(h_StackE, d_StackE, totalSize * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_StackB); // release temperary variable
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "*** FAILED - ABORTING: initial image preparation failed \n");
		exit(1);
	}
	// Create FFT plans
	hipfftPlan3d(&fftPlanFwd, sx, sy, sz, HIPFFT_R2C);
	hipfftPlan3d(&fftPlanInv, sx, sy, sz, HIPFFT_C2R);
	hipMemGetInfo(&freeMem, &totalMem);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "*** FAILED - ABORTING: cufftPlan error \n");
		exit(1);
	}
	printf("...GPU free memory (before decon iteration) is %.0f MBites\n", (float)freeMem / 1048576.0f);
	for (int itNum = 1; itNum <= itNumForDecon; itNum++) {
		//printf("...Processing iteration %d\n", it);
		// ### 1st view
		hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_StackE, (hipfftComplex *)d_StackESpectrum);
		hipMemcpy(h_StackE, d_StackE, totalSize * sizeof(float), hipMemcpyDeviceToHost);

		hipMemcpy(d_OTF, h_OTF1, totalSizeSpectrum * sizeof(fComplex), hipMemcpyHostToDevice);
		multicomplex3Dgpu(d_StackESpectrum, d_StackESpectrum, d_OTF, sx, sy, (sz / 2 + 1));
		hipfftExecC2R(fftPlanInv, (hipfftComplex *)d_StackESpectrum, (hipfftReal *)d_StackE);

		hipMemcpy(d_StackA, h_StackA, totalSize * sizeof(float), hipMemcpyHostToDevice);
		div3Dgpu(d_StackE, d_StackA, d_StackE, sx, sy, sz);   
															
		hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_StackE, (hipfftComplex *)d_StackESpectrum);
		hipMemcpy(d_OTF, h_OTF_bp1, totalSizeSpectrum * sizeof(fComplex), hipMemcpyHostToDevice);
		multicomplex3Dgpu(d_StackESpectrum, d_StackESpectrum, d_OTF, sx, sy, (sz / 2 + 1));
		hipfftExecC2R(fftPlanInv, (hipfftComplex *)d_StackESpectrum, (hipfftReal *)d_StackE);
																				
		hipMemcpy(d_StackA, h_StackE, totalSize * sizeof(float), hipMemcpyHostToDevice);
		multi3Dgpu(d_StackE, d_StackE, d_StackA, sx, sy, sz);//
		maxvalue3Dgpu(d_StackE, d_StackE, float(SMALLVALUE), sx, sy, sz);

		// ### 2nd view	
		hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_StackE, (hipfftComplex *)d_StackESpectrum);//
		hipMemcpy(h_StackE, d_StackE, totalSize * sizeof(float), hipMemcpyDeviceToHost);

		hipMemcpy(d_OTF, h_OTF2, totalSizeSpectrum * sizeof(fComplex), hipMemcpyHostToDevice);
		multicomplex3Dgpu(d_StackESpectrum, d_StackESpectrum, d_OTF, sx, sy, (sz / 2 + 1));
		hipfftExecC2R(fftPlanInv, (hipfftComplex *)d_StackESpectrum, (hipfftReal *)d_StackE);
	
		hipMemcpy(d_StackA, h_StackB, totalSize * sizeof(float), hipMemcpyHostToDevice);
		div3Dgpu(d_StackE, d_StackA, d_StackE, sx, sy, sz);//

		hipfftExecR2C(fftPlanFwd, (hipfftReal *)d_StackE, (hipfftComplex *)d_StackESpectrum);
		hipMemcpy(d_OTF, h_OTF_bp2, totalSizeSpectrum * sizeof(fComplex), hipMemcpyHostToDevice);
		multicomplex3Dgpu(d_StackESpectrum, d_StackESpectrum, d_OTF, sx, sy, (sz / 2 + 1));
		hipfftExecC2R(fftPlanInv, (hipfftComplex *)d_StackESpectrum, (hipfftReal *)d_StackE);

		hipMemcpy(d_StackA, h_StackE, totalSize * sizeof(float), hipMemcpyHostToDevice);
		multi3Dgpu(d_StackE, d_StackE, d_StackA, sx, sy, sz);
		maxvalue3Dgpu(d_StackE, d_StackE, float(SMALLVALUE), sx, sy, sz);
	}
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "*** FAILED - ABORTING: decon iterration error \n");
		exit(1);
	}
	free(h_StackA); free(h_StackB); free(h_StackE);
	hipfftDestroy(fftPlanFwd);
	hipfftDestroy(fftPlanInv);
	hipMemGetInfo(&freeMem, &totalMem);
	printf("...GPU free memory (after decon iteration) is %.0f MBites\n", (float)freeMem / 1048576.0f);
	end = clock();
	printf("...Time cost for decon is %2.3f s\n", (float)(end - start) / CLOCKS_PER_SEC);
	return 0;
}

#undef SMALLVALUE
#undef NDIM